//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: visionPipeline_terminate.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 24-Feb-2023 21:05:52
//

// Include Files
#include "visionPipeline_terminate.h"
#include "visionPipeline_data.h"
#include "MWMemoryManager.hpp"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void visionPipeline_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorString(errCode),
            hipGetErrorName(errCode));
    exit(errCode);
  }
  mwMemoryManagerTerminate();
  isInitialized_gpuMEX = false;
}

//
// File trailer for visionPipeline_terminate.cu
//
// [EOF]
//
