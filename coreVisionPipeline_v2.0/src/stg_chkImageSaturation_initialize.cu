//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: stg_chkImageSaturation_initialize.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 24-Feb-2023 22:32:24
//

// Include Files
#include "stg_chkImageSaturation_initialize.h"
#include "stg_chkImageSaturation_data.h"
#include "MWMemoryManager.hpp"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void stg_chkImageSaturation_initialize()
{
  mwMemoryManagerInit(256U, 0U, 8U, 2048U);
  hipGetLastError();
  isInitialized_stg_chkImageSaturation = true;
}

//
// File trailer for stg_chkImageSaturation_initialize.cu
//
// [EOF]
//
