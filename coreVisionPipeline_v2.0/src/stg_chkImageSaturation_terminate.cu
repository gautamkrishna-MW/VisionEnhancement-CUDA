//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: stg_chkImageSaturation_terminate.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 24-Feb-2023 22:32:24
//

// Include Files
#include "stg_chkImageSaturation_terminate.h"
#include "stg_chkImageSaturation_data.h"
#include "MWMemoryManager.hpp"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void stg_chkImageSaturation_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorString(errCode),
            hipGetErrorName(errCode));
    exit(errCode);
  }
  mwMemoryManagerTerminate();
  isInitialized_stg_chkImageSaturation = false;
}

//
// File trailer for stg_chkImageSaturation_terminate.cu
//
// [EOF]
//
