#include "hip/hip_runtime.h"
//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: stg_chkImageSaturation.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 24-Feb-2023 22:32:24
//

// Include Files
#include "stg_chkImageSaturation.h"
#include "stg_chkImageSaturation_data.h"
#include "stg_chkImageSaturation_initialize.h"
#include "MWAtomicUtility.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"

// Function Declarations
static __global__ void
stg_chkImageSaturation_kernel1(unsigned long long localHistogram[16384]);

static __global__ void
stg_chkImageSaturation_kernel2(const unsigned short inpFrame[2108160],
                               unsigned long long localHistogram[16384]);

static __global__ void
stg_chkImageSaturation_kernel3(unsigned long long globalHistogram[4096]);

static __global__ void
stg_chkImageSaturation_kernel4(unsigned long long localHistogram[16384],
                               unsigned long long globalHistogram[4096]);

static __global__ void
stg_chkImageSaturation_kernel5(const unsigned long long globalHistogram[4096],
                               float *numPixels);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long localHistogram[16384]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void stg_chkImageSaturation_kernel1(
    unsigned long long localHistogram[16384])
{
  unsigned long long threadId;
  int histIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  histIter = static_cast<int>(threadId);
  if (histIter < 16384) {
    //  Compute sizes and declaring constants
    //  Histogram Computation
    //  GPU Codegen: Block-wise histogram computation
    //  Histogram per block is stored in the local histogram matrix
    localHistogram[histIter] = 0ULL;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short inpFrame[2108160]
//                unsigned long long localHistogram[16384]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void stg_chkImageSaturation_kernel2(
    const unsigned short inpFrame[2108160],
    unsigned long long localHistogram[16384])
{
  unsigned long long threadId;
  int colBlockIter;
  int colIter;
  int rowBlockIter;
  int rowIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int>(threadId % 549ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowIter)) / 549ULL;
  colIter = static_cast<int>(threadId % 960ULL);
  threadId = (threadId - static_cast<unsigned long long>(colIter)) / 960ULL;
  rowBlockIter = static_cast<int>(threadId % 2ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowBlockIter)) / 2ULL;
  colBlockIter = static_cast<int>(threadId);
  if ((colBlockIter < 2) && (rowBlockIter < 2) && (colIter < 960) &&
      (rowIter < 549)) {
    unsigned int u;
    colBlockIter = colBlockIter * 960 + 1;
    rowBlockIter = rowBlockIter * 549 + 1;
    u = inpFrame[((rowBlockIter + rowIter) +
                  1098 * ((colBlockIter + colIter) - 1)) -
                 1] +
        1U;
    if (u > 65535U) {
      u = 65535U;
    }
    gpu_uint64_atomicAdd(
        &localHistogram
            [((static_cast<int>(u) +
               ((static_cast<int>(
                     (static_cast<double>(rowBlockIter) - 1.0) / 549.0 + 1.0) -
                 1)
                << 12)) +
              ((static_cast<int>(
                    (static_cast<double>(colBlockIter) - 1.0) / 960.0 + 1.0) -
                1)
               << 13)) -
             1],
        1ULL);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long globalHistogram[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void stg_chkImageSaturation_kernel3(
    unsigned long long globalHistogram[4096])
{
  unsigned long long threadId;
  int histIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  histIter = static_cast<int>(threadId);
  if (histIter < 4096) {
    //  Local histograms are added to create the final global histogram
    globalHistogram[histIter] = 0ULL;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long localHistogram[16384]
//                unsigned long long globalHistogram[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void stg_chkImageSaturation_kernel4(
    unsigned long long localHistogram[16384],
    unsigned long long globalHistogram[4096])
{
  unsigned long long threadId;
  int colBlockIter;
  int histIter;
  int rowBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  histIter = static_cast<int>(threadId % 4096ULL);
  threadId = (threadId - static_cast<unsigned long long>(histIter)) / 4096ULL;
  rowBlockIter = static_cast<int>(threadId % 2ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowBlockIter)) / 2ULL;
  colBlockIter = static_cast<int>(threadId);
  if ((colBlockIter < 2) && (rowBlockIter < 2) && (histIter < 4096)) {
    gpu_uint64_atomicAdd(&globalHistogram[histIter],
                         localHistogram[(histIter + (rowBlockIter << 12)) +
                                        (colBlockIter << 13)]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned long long globalHistogram[4096]
//                float *numPixels
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void stg_chkImageSaturation_kernel5(
    const unsigned long long globalHistogram[4096], float *numPixels)
{
  unsigned long long threadId;
  int i;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 2) {
    //  Dummy Kernel call
    for (i = 0; i < 5; i++) {
      *numPixels += static_cast<float>(globalHistogram[i + 4091]);
    }
  }
}

//
// GPU Pragmas
//
// Arguments    : const unsigned short inpFrame[2108160]
// Return Type  : boolean_T
//
boolean_T stg_chkImageSaturation(const unsigned short inpFrame[2108160])
{
  unsigned long long(*gpu_localHistogram)[16384];
  unsigned long long(*gpu_globalHistogram)[4096];
  float numPixels;
  float *gpu_numPixels;
  unsigned short(*gpu_inpFrame)[2108160];
  boolean_T isSaturated;
  if (!isInitialized_stg_chkImageSaturation) {
    stg_chkImageSaturation_initialize();
  }
  mwCudaMalloc(&gpu_numPixels, 4ULL);
  mwCudaMalloc(&gpu_globalHistogram, 32768ULL);
  mwCudaMalloc(&gpu_inpFrame, 4216320ULL);
  mwCudaMalloc(&gpu_localHistogram, 131072ULL);
  //  Compute sizes and declaring constants
  //  Histogram Computation
  //  GPU Codegen: Block-wise histogram computation
  //  Histogram per block is stored in the local histogram matrix
  stg_chkImageSaturation_kernel1<<<dim3(32U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_localHistogram);
  hipMemcpy(*gpu_inpFrame, inpFrame, 4216320ULL, hipMemcpyHostToDevice);
  stg_chkImageSaturation_kernel2<<<dim3(4118U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_inpFrame, *gpu_localHistogram);
  //  Local histograms are added to create the final global histogram
  stg_chkImageSaturation_kernel3<<<dim3(8U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_globalHistogram);
  stg_chkImageSaturation_kernel4<<<dim3(32U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_localHistogram, *gpu_globalHistogram);
  //  Check of the pixels are concentrated in the last K bins
  numPixels = 0.0F;
  hipMemcpy(gpu_numPixels, &numPixels, 4ULL, hipMemcpyHostToDevice);
  stg_chkImageSaturation_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_globalHistogram, gpu_numPixels);
  hipMemcpy(&numPixels, gpu_numPixels, 4ULL, hipMemcpyDeviceToHost);
  isSaturated = (numPixels / 2.10816E+6F > 0.9);
  mwCudaFree(&(*gpu_localHistogram)[0]);
  mwCudaFree(&(*gpu_inpFrame)[0]);
  mwCudaFree(&(*gpu_globalHistogram)[0]);
  mwCudaFree(gpu_numPixels);
  return isSaturated;
}

//
// File trailer for stg_chkImageSaturation.cu
//
// [EOF]
//
