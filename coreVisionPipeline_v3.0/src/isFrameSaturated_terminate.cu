//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: isFrameSaturated_terminate.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 13-Mar-2023 12:15:40
//

// Include Files
#include "isFrameSaturated_terminate.h"
#include "isFrameSaturated_data.h"
#include "MWMemoryManager.hpp"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void isFrameSaturated_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorString(errCode),
            hipGetErrorName(errCode));
    exit(errCode);
  }
  mwMemoryManagerTerminate();
  isInitialized_isFrameSaturated = false;
}

//
// File trailer for isFrameSaturated_terminate.cu
//
// [EOF]
//
