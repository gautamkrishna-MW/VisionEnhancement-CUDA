//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: coreVisionPipeline_terminate.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 17-Mar-2023 21:49:01
//

// Include Files
#include "coreVisionPipeline_terminate.h"
#include "coreVisionPipeline_data.h"
#include "MWMemoryManager.hpp"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void coreVisionPipeline_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorString(errCode),
            hipGetErrorName(errCode));
    exit(errCode);
  }
  mwMemoryManagerTerminate();
  isInitialized_coreVisionPipeline = false;
}

//
// File trailer for coreVisionPipeline_terminate.cu
//
// [EOF]
//
