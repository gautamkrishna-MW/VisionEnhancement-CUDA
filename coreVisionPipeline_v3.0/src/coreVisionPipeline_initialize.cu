//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: coreVisionPipeline_initialize.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 17-Mar-2023 21:49:01
//

// Include Files
#include "coreVisionPipeline_initialize.h"
#include "coreVisionPipeline_data.h"
#include "MWMemoryManager.hpp"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void coreVisionPipeline_initialize()
{
  mwMemoryManagerInit(256U, 0U, 8U, 2048U);
  hipGetLastError();
  isInitialized_coreVisionPipeline = true;
}

//
// File trailer for coreVisionPipeline_initialize.cu
//
// [EOF]
//
