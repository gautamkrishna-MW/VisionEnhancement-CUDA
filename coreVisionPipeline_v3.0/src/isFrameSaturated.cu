#include "hip/hip_runtime.h"
//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: isFrameSaturated.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 13-Mar-2023 12:15:40
//

// Include Files
#include "isFrameSaturated.h"
#include "isFrameSaturated_data.h"
#include "isFrameSaturated_initialize.h"
#include "MWAtomicUtility.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"

// Function Declarations
static void checkCudaError(hipError_t errCode, const char *file,
                           unsigned int b_line);

static __global__ void
isFrameSaturated_kernel1(unsigned long long localHistogram[16384]);

static __global__ void
isFrameSaturated_kernel2(const unsigned short inpFrame[2108160],
                         unsigned long long localHistogram[16384]);

static __global__ void
isFrameSaturated_kernel3(unsigned long long globalHistogram[4096]);

static __global__ void
isFrameSaturated_kernel4(unsigned long long localHistogram[16384],
                         unsigned long long globalHistogram[4096]);

static __global__ void
isFrameSaturated_kernel5(const unsigned long long globalHistogram[4096],
                         unsigned long long cumulativePixDist[4096]);

static __global__ void
isFrameSaturated_kernel6(const unsigned long long cumulativePixDist[4096],
                         boolean_T *isSaturated);

static void raiseCudaError(int errCode, const char *file, unsigned int b_line,
                           const char *errorName, const char *errorString);

// Function Definitions
//
// Arguments    : hipError_t errCode
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
static void checkCudaError(hipError_t errCode, const char *file,
                           unsigned int b_line)
{
  if (errCode != hipSuccess) {
    raiseCudaError(errCode, file, b_line, hipGetErrorString(errCode),
                   hipGetErrorName(errCode));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long localHistogram[16384]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void isFrameSaturated_kernel1(
    unsigned long long localHistogram[16384])
{
  unsigned long long threadId;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId);
  if (k < 16384) {
    //  Compute sizes and declaring constants
    //  Histogram Computation
    //  GPU Codegen: Block-wise histogram computation
    //  Histogram per block is stored in the local histogram matrix
    localHistogram[k] = 0ULL;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short inpFrame[2108160]
//                unsigned long long localHistogram[16384]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void isFrameSaturated_kernel2(
    const unsigned short inpFrame[2108160],
    unsigned long long localHistogram[16384])
{
  unsigned long long threadId;
  int colBlockIter;
  int rowBlockIter;
  int shiftAmount;
  int xexp;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  shiftAmount = static_cast<int>(threadId % 549ULL);
  threadId = (threadId - static_cast<unsigned long long>(shiftAmount)) / 549ULL;
  xexp = static_cast<int>(threadId % 960ULL);
  threadId = (threadId - static_cast<unsigned long long>(xexp)) / 960ULL;
  rowBlockIter = static_cast<int>(threadId % 2ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowBlockIter)) / 2ULL;
  colBlockIter = static_cast<int>(threadId);
  if ((colBlockIter < 2) && (rowBlockIter < 2) && (xexp < 960) &&
      (shiftAmount < 549)) {
    unsigned int u;
    colBlockIter = colBlockIter * 960 + 1;
    rowBlockIter = rowBlockIter * 549 + 1;
    u = inpFrame[((rowBlockIter + shiftAmount) +
                  1098 * ((colBlockIter + xexp) - 1)) -
                 1] +
        1U;
    if (u > 65535U) {
      u = 65535U;
    }
    gpu_uint64_atomicAdd(
        &localHistogram
            [((static_cast<int>(u) +
               ((static_cast<int>(
                     (static_cast<double>(rowBlockIter) - 1.0) / 549.0 + 1.0) -
                 1)
                << 12)) +
              ((static_cast<int>(
                    (static_cast<double>(colBlockIter) - 1.0) / 960.0 + 1.0) -
                1)
               << 13)) -
             1],
        1ULL);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long globalHistogram[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void isFrameSaturated_kernel3(
    unsigned long long globalHistogram[4096])
{
  unsigned long long threadId;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId);
  if (k < 4096) {
    //  Local histograms are added to create the final global histogram
    globalHistogram[k] = 0ULL;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long localHistogram[16384]
//                unsigned long long globalHistogram[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void isFrameSaturated_kernel4(
    unsigned long long localHistogram[16384],
    unsigned long long globalHistogram[4096])
{
  unsigned long long threadId;
  int colBlockIter;
  int k;
  int rowBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId % 4096ULL);
  threadId = (threadId - static_cast<unsigned long long>(k)) / 4096ULL;
  rowBlockIter = static_cast<int>(threadId % 2ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowBlockIter)) / 2ULL;
  colBlockIter = static_cast<int>(threadId);
  if ((colBlockIter < 2) && (rowBlockIter < 2) && (k < 4096)) {
    gpu_uint64_atomicAdd(
        &globalHistogram[k],
        localHistogram[(k + (rowBlockIter << 12)) + (colBlockIter << 13)]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned long long globalHistogram[4096]
//                unsigned long long cumulativePixDist[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void isFrameSaturated_kernel5(
    const unsigned long long globalHistogram[4096],
    unsigned long long cumulativePixDist[4096])
{
  unsigned long long threadId;
  int k;
  int xexp;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId);
  if (k < 4096) {
    threadId = globalHistogram[k];
    if (threadId == 0ULL) {
      cumulativePixDist[k] = 0ULL;
    } else {
      unsigned long long res;
      frexp(2.10816E+6, &xexp);
      xexp = -31;
      res = threadId / 4527239127367680ULL;
      threadId -= threadId / 4527239127367680ULL * 4527239127367680ULL;
      int exitg1;
      do {
        exitg1 = 0;
        if (xexp < 0) {
          int shiftAmount;
          shiftAmount = -xexp;
          if (shiftAmount > 11) {
            shiftAmount = 11;
          }
          if ((res >> (64 - shiftAmount)) > 0ULL) {
            cumulativePixDist[k] = MAX_uint64_T;
            exitg1 = 1;
          } else {
            unsigned long long t;
            res <<= shiftAmount;
            threadId <<= shiftAmount;
            xexp += shiftAmount;
            t = threadId / 4527239127367680ULL;
            if (MAX_uint64_T - t <= res) {
              cumulativePixDist[k] = MAX_uint64_T;
              exitg1 = 1;
            } else {
              res += t;
              threadId -= threadId / 4527239127367680ULL * 4527239127367680ULL;
            }
          }
        } else {
          if ((threadId << 1) >= 4527239127367680ULL) {
            res++;
          }
          cumulativePixDist[k] = res;
          exitg1 = 1;
        }
      } while (exitg1 == 0);
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned long long cumulativePixDist[4096]
//                boolean_T *isSaturated
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void isFrameSaturated_kernel6(
    const unsigned long long cumulativePixDist[4096], boolean_T *isSaturated)
{
  unsigned long long threadId;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId);
  if (k < 4095) {
    unsigned long long res;
    threadId = cumulativePixDist[k];
    res = threadId + cumulativePixDist[k + 1];
    if (res < threadId) {
      res = MAX_uint64_T;
    }
    *isSaturated =
        ((res >= 4503599627370496ULL) || (res > 0.9) || (*isSaturated));
  }
}

//
// Arguments    : int errCode
//                const char *file
//                unsigned int b_line
//                const char *errorName
//                const char *errorString
// Return Type  : void
//
static void raiseCudaError(int errCode, const char *file, unsigned int b_line,
                           const char *errorName, const char *errorString)
{
  printf("ERR[%d] %s:%s in file %s at line %d\nExiting program execution ...\n",
         errCode, errorName, errorString, file, b_line);
  exit(errCode);
}

//
// GPU Pragmas
//
// Arguments    : const unsigned short inpFrame[2108160]
// Return Type  : boolean_T
//
boolean_T isFrameSaturated(const unsigned short inpFrame[2108160])
{
  unsigned long long(*gpu_localHistogram)[16384];
  unsigned long long(*gpu_cumulativePixDist)[4096];
  unsigned long long(*gpu_globalHistogram)[4096];
  boolean_T isSaturated;
  boolean_T *gpu_isSaturated;
  if (!isInitialized_isFrameSaturated) {
    isFrameSaturated_initialize();
  }
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  mwCudaMalloc(&gpu_isSaturated, 1ULL);
  CUDACHECK(hipGetLastError());
  mwCudaMalloc(&gpu_cumulativePixDist, 32768ULL);
  CUDACHECK(hipGetLastError());
  mwCudaMalloc(&gpu_globalHistogram, 32768ULL);
  CUDACHECK(hipGetLastError());
  mwCudaMalloc(&gpu_localHistogram, 131072ULL);
  CUDACHECK(hipGetLastError());
  //  Compute sizes and declaring constants
  //  Histogram Computation
  //  GPU Codegen: Block-wise histogram computation
  //  Histogram per block is stored in the local histogram matrix
  isFrameSaturated_kernel1<<<dim3(32U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_localHistogram);
  CUDACHECK(hipGetLastError());
  isFrameSaturated_kernel2<<<dim3(4118U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      inpFrame, *gpu_localHistogram);
  CUDACHECK(hipGetLastError());
  //  Local histograms are added to create the final global histogram
  isFrameSaturated_kernel3<<<dim3(8U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_globalHistogram);
  CUDACHECK(hipGetLastError());
  isFrameSaturated_kernel4<<<dim3(32U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_localHistogram, *gpu_globalHistogram);
  CUDACHECK(hipGetLastError());
  //  Check of the pixels are concentrated in the last K bins
  callThrustScan1D(&(*gpu_globalHistogram)[0], false, 4096);
  isFrameSaturated_kernel5<<<dim3(8U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_globalHistogram, *gpu_cumulativePixDist);
  CUDACHECK(hipGetLastError());
  isSaturated = false;
  hipMemcpy(gpu_isSaturated, &isSaturated, 1ULL, hipMemcpyHostToDevice);
  CUDACHECK(hipGetLastError());
  isFrameSaturated_kernel6<<<dim3(8U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_cumulativePixDist, gpu_isSaturated);
  CUDACHECK(hipGetLastError());
  hipMemcpy(&isSaturated, gpu_isSaturated, 1ULL, hipMemcpyDeviceToHost);
  CUDACHECK(hipGetLastError());
  mwCudaFree(&(*gpu_localHistogram)[0]);
  CUDACHECK(hipGetLastError());
  mwCudaFree(&(*gpu_globalHistogram)[0]);
  CUDACHECK(hipGetLastError());
  mwCudaFree(&(*gpu_cumulativePixDist)[0]);
  CUDACHECK(hipGetLastError());
  mwCudaFree(gpu_isSaturated);
  CUDACHECK(hipGetLastError());
#undef CUDACHECK
  return isSaturated;
}

//
// File trailer for isFrameSaturated.cu
//
// [EOF]
//
