#include "hip/hip_runtime.h"
//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: coreVisionPipeline.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 17-Mar-2023 21:49:01
//

// Include Files
#include "coreVisionPipeline.h"
#include "coreVisionPipeline_data.h"
#include "coreVisionPipeline_initialize.h"
#include "MWAtomicUtility.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"
#include "MWTransposeUtility.hpp"
#include "thrust/sort.h"

// Function Declarations
static __device__ float atomicOpreal32_T(float *address, float value);

static __device__ unsigned int atomicOpuint32_T(unsigned int *address,
                                                unsigned int value);

static __global__ void coder_reduce0(const float inputVar[527040],
                                     float *outputVar);

static __global__ void coder_reduce1(const float inputVar[527040],
                                     float *outputVar);

static __global__ void coder_reduce2(const float inputVar[527040],
                                     float *outputVar);

static __global__ void coder_reduce3(const unsigned short inputVar[19200],
                                     unsigned int *outputVar);

static __global__ void
coreVisionPipeline_kernel1(const unsigned short inputFrame[2108160],
                           float gMat[527040], float bMat[527040],
                           float rMat[527040]);

static __global__ void
coreVisionPipeline_kernel10(const unsigned short varargout_4[2073600],
                            const unsigned short varargout_5[2073600],
                            const unsigned short varargout_3[2073600],
                            const unsigned short varargout_2[2073600],
                            const unsigned short varargout_1[2073600],
                            unsigned short stg2OutFrame[6220800]);

static __global__ void
coreVisionPipeline_kernel11(const unsigned short stg2OutFrame[6220800],
                            unsigned short stg3OutFrame[6220800]);

static __global__ void
coreVisionPipeline_kernel12(const unsigned short stg2OutFrame[6220800],
                            unsigned short stg3OutFrame[6220800]);

static __global__ void
coreVisionPipeline_kernel13(const float gainAWB[3],
                            unsigned short stg1OutFrame[2073600],
                            unsigned short stg3OutFrame[6220800]);

static __global__ void
coreVisionPipeline_kernel14(unsigned long long localHistogram[16384]);

static __global__ void
coreVisionPipeline_kernel15(const unsigned short stg1OutFrame[2073600],
                            unsigned long long localHistogram[16384]);

static __global__ void
coreVisionPipeline_kernel16(unsigned long long globalHistogram[4096]);

static __global__ void
coreVisionPipeline_kernel17(unsigned long long localHistogram[16384],
                            unsigned long long globalHistogram[4096]);

static __global__ void
coreVisionPipeline_kernel18(const unsigned long long globalHistogram[4096],
                            int *bin99Percent, int *bin1Percent);

static __global__ void
coreVisionPipeline_kernel19(const int *bin99Percent, const int *bin1Percent,
                            unsigned short stg3OutFrame[6220800]);

static __global__ void coreVisionPipeline_kernel2(const float rMat[527040],
                                                  float *b);

static __global__ void
coreVisionPipeline_kernel20(const unsigned short stg3OutFrame[6220800],
                            unsigned short stg1OutFrame[2073600]);

static __global__ void
coreVisionPipeline_kernel21(const unsigned short stg3OutFrame[6220800],
                            unsigned short stg1OutFrame[2073600]);

static __global__ void
coreVisionPipeline_kernel22(const unsigned short stg3OutFrame[6220800],
                            unsigned short stg1OutFrame[2073600]);

static __global__ void
coreVisionPipeline_kernel23(const unsigned short outImgB[2073600],
                            const unsigned short outImgG[2073600],
                            const unsigned short outImgR[2073600],
                            unsigned short processedFrame[6220800]);

static __global__ void coreVisionPipeline_kernel3(const float gMat[527040],
                                                  float *b);

static __global__ void coreVisionPipeline_kernel4(const float bMat[527040],
                                                  float *b);

static __global__ void coreVisionPipeline_kernel5(float gainAWB[3]);

static __global__ void coreVisionPipeline_kernel6(const float meanBChannel,
                                                  const float meanGChannel,
                                                  const float meanRChannel,
                                                  float gainAWB[3]);

static __global__ void
coreVisionPipeline_kernel7(const unsigned short outFrameColMajor[2108160],
                           unsigned short inputArray[19200]);

static __global__ void
coreVisionPipeline_kernel8(const unsigned short inputArray[19200],
                           unsigned int *outputVar);

static __global__ void
coreVisionPipeline_kernel9(const float meanRChannel,
                           const unsigned short outFrameColMajor[2108160],
                           unsigned short stg1OutFrame[2073600]);

static __device__ float shflDown1(float in1, unsigned int offset,
                                  unsigned int mask);

static __device__ unsigned int shflDown1(unsigned int in1, unsigned int offset,
                                         unsigned int mask);

static __global__ void stencilKernel(const unsigned short input[2073600],
                                     unsigned short paddingValue,
                                     unsigned short output[2073600],
                                     unsigned short b_output[2073600],
                                     unsigned short c_output[2073600],
                                     unsigned short d_output[2073600],
                                     unsigned short e_output[2073600]);

static __device__ float threadGroupReduction(float val, unsigned int lane,
                                             unsigned int mask);

static __device__ unsigned int
threadGroupReduction(unsigned int val, unsigned int lane, unsigned int mask);

static __device__ unsigned int workGroupReduction(unsigned int val,
                                                  unsigned int mask,
                                                  unsigned int numActiveWarps);

static __device__ float workGroupReduction(float val, unsigned int mask,
                                           unsigned int numActiveWarps);

// Function Definitions
//
// Arguments    : float *address
//                float value
// Return Type  : float
//
static __device__ float atomicOpreal32_T(float *address, float value)
{
  unsigned int old;
  unsigned int *address_as_up;
  address_as_up = (unsigned int *)address;
  old = *address_as_up;
  float input2;
  unsigned int assumed;
  do {
    assumed = old;
    input2 = __uint_as_float(old);
    //  Helper function
    old = atomicCAS(address_as_up, old, __float_as_uint(value + input2));
  } while (assumed != old);
  return __uint_as_float(old);
}

//
// Arguments    : unsigned int *address
//                unsigned int value
// Return Type  : unsigned int
//
static __device__ unsigned int atomicOpuint32_T(unsigned int *address,
                                                unsigned int value)
{
  unsigned int output;
  output = *address;
  unsigned int assumed;
  unsigned int red;
  do {
    assumed = output;
    //  Helper function
    red = value + output;
    if (red < value) {
      red = MAX_uint32_T;
    }
    output = atomicCAS(address, output, red);
  } while (assumed != output);
  return output;
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float inputVar[527040]
//                float *outputVar
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void coder_reduce0(const float inputVar[527040],
                                                  float *outputVar)
{
  float input1;
  unsigned int blockStride;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  if (static_cast<unsigned int>(mwGetBlockIndex()) == 527039U / blockStride) {
    m = 527039U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 527038U) {
    input1 = inputVar[threadId];
  }
  m = __ballot_sync(MAX_uint32_T, threadId <= 527038U);
  for (unsigned int idx{threadId + threadStride}; idx <= 527038U;
       idx += threadStride) {
    float input2;
    input2 = inputVar[idx];
    //  Helper function
    input1 += input2;
  }
  input1 = workGroupReduction(input1, m, blockStride);
  if ((threadId <= 527038U) && (thBlkId == 0U)) {
    atomicOpreal32_T(&outputVar[0], input1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float inputVar[527040]
//                float *outputVar
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void coder_reduce1(const float inputVar[527040],
                                                  float *outputVar)
{
  float input1;
  unsigned int blockStride;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  if (static_cast<unsigned int>(mwGetBlockIndex()) == 527039U / blockStride) {
    m = 527039U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 527038U) {
    input1 = inputVar[threadId];
  }
  m = __ballot_sync(MAX_uint32_T, threadId <= 527038U);
  for (unsigned int idx{threadId + threadStride}; idx <= 527038U;
       idx += threadStride) {
    float input2;
    input2 = inputVar[idx];
    //  Helper function
    input1 += input2;
  }
  input1 = workGroupReduction(input1, m, blockStride);
  if ((threadId <= 527038U) && (thBlkId == 0U)) {
    atomicOpreal32_T(&outputVar[0], input1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float inputVar[527040]
//                float *outputVar
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void coder_reduce2(const float inputVar[527040],
                                                  float *outputVar)
{
  float input1;
  unsigned int blockStride;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  if (static_cast<unsigned int>(mwGetBlockIndex()) == 527039U / blockStride) {
    m = 527039U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 527038U) {
    input1 = inputVar[threadId];
  }
  m = __ballot_sync(MAX_uint32_T, threadId <= 527038U);
  for (unsigned int idx{threadId + threadStride}; idx <= 527038U;
       idx += threadStride) {
    float input2;
    input2 = inputVar[idx];
    //  Helper function
    input1 += input2;
  }
  input1 = workGroupReduction(input1, m, blockStride);
  if ((threadId <= 527038U) && (thBlkId == 0U)) {
    atomicOpreal32_T(&outputVar[0], input1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short inputVar[19200]
//                unsigned int *outputVar
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void coder_reduce3(
    const unsigned short inputVar[19200], unsigned int *outputVar)
{
  unsigned int blockStride;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  unsigned int tmpRed0;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  if (static_cast<unsigned int>(mwGetBlockIndex()) == 19199U / blockStride) {
    m = 19199U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 19198U) {
    unsigned short input1;
    input1 = inputVar[threadId];
    tmpRed0 = input1;
  }
  m = __ballot_sync(MAX_uint32_T, threadId <= 19198U);
  for (unsigned int idx{threadId + threadStride}; idx <= 19198U;
       idx += threadStride) {
    unsigned int b_input1;
    int input2;
    b_input1 = tmpRed0;
    input2 = inputVar[idx];
    //  Helper function
    tmpRed0 += static_cast<unsigned int>(input2);
    if (tmpRed0 < b_input1) {
      tmpRed0 = MAX_uint32_T;
    }
  }
  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if ((threadId <= 19198U) && (thBlkId == 0U)) {
    atomicOpuint32_T(&outputVar[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short inputFrame[2108160]
//                float gMat[527040]
//                float bMat[527040]
//                float rMat[527040]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel1(
    const unsigned short inputFrame[2108160], float gMat[527040],
    float bMat[527040], float rMat[527040])
{
  unsigned long long threadId;
  int colIter;
  int rowIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int>(threadId % 275ULL);
  colIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(rowIter)) / 275ULL);
  if ((colIter < 481) && (rowIter < 275)) {
    double histCol;
    int colBlockIter;
    unsigned int qY;
    colBlockIter = (colIter << 1) + 480;
    histCol = static_cast<double>(rowIter) * 2.0 + 274.5;
    colIter = static_cast<int>(floor((histCol - 274.5) / 2.0));
    rowIter = static_cast<int>(
        floor((static_cast<double>(colBlockIter) - 480.0) / 2.0));
    rMat[colIter + 549 * rowIter] = inputFrame
        [(static_cast<int>(floor(histCol)) + 1098 * (colBlockIter - 1)) - 1];
    bMat[colIter + 549 * rowIter] =
        inputFrame[static_cast<int>(floor(histCol)) + 1098 * colBlockIter];
    qY =
        static_cast<unsigned int>(inputFrame[(static_cast<int>(floor(histCol)) +
                                              1098 * colBlockIter) -
                                             1]) +
        inputFrame[static_cast<int>(floor(histCol)) +
                   1098 * (colBlockIter - 1)];
    if (qY > 65535U) {
      qY = 65535U;
    }
    gMat[colIter + 549 * rowIter] = static_cast<float>(qY);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short varargout_4[2073600]
//                const unsigned short varargout_5[2073600]
//                const unsigned short varargout_3[2073600]
//                const unsigned short varargout_2[2073600]
//                const unsigned short varargout_1[2073600]
//                unsigned short stg2OutFrame[6220800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel10(
    const unsigned short varargout_4[2073600],
    const unsigned short varargout_5[2073600],
    const unsigned short varargout_3[2073600],
    const unsigned short varargout_2[2073600],
    const unsigned short varargout_1[2073600],
    unsigned short stg2OutFrame[6220800])
{
  unsigned long long threadId;
  int chIter;
  int colIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  chIter = static_cast<int>(threadId % 540ULL);
  colIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(chIter)) / 540ULL);
  if ((colIter < 960) && (chIter < 540)) {
    int colBlockIter;
    int rowIter;
    colBlockIter = (colIter << 1) + 1;
    rowIter = (chIter << 1) + 1;
    stg2OutFrame[(rowIter + 1080 * (colBlockIter - 1)) - 1] =
        varargout_1[(rowIter + 1080 * (colBlockIter - 1)) - 1];
    stg2OutFrame[(rowIter + 1080 * (colBlockIter - 1)) + 2073599] =
        varargout_2[(rowIter + 1080 * (colBlockIter - 1)) - 1];
    stg2OutFrame[(rowIter + 1080 * (colBlockIter - 1)) + 4147199] =
        varargout_3[(rowIter + 1080 * (colBlockIter - 1)) - 1];
    stg2OutFrame[rowIter + 1080 * (colBlockIter - 1)] =
        varargout_5[rowIter + 1080 * (colBlockIter - 1)];
    stg2OutFrame[(rowIter + 1080 * (colBlockIter - 1)) + 2073600] =
        varargout_1[rowIter + 1080 * (colBlockIter - 1)];
    stg2OutFrame[(rowIter + 1080 * (colBlockIter - 1)) + 4147200] =
        varargout_4[rowIter + 1080 * (colBlockIter - 1)];
    colBlockIter = (colIter << 1) + 1;
    rowIter = (chIter << 1) + 1;
    stg2OutFrame[(rowIter + 1080 * colBlockIter) - 1] =
        varargout_4[(rowIter + 1080 * colBlockIter) - 1];
    stg2OutFrame[(rowIter + 1080 * colBlockIter) + 2073599] =
        varargout_1[(rowIter + 1080 * colBlockIter) - 1];
    stg2OutFrame[(rowIter + 1080 * colBlockIter) + 4147199] =
        varargout_5[(rowIter + 1080 * colBlockIter) - 1];
    stg2OutFrame[rowIter + 1080 * colBlockIter] =
        varargout_3[rowIter + 1080 * colBlockIter];
    stg2OutFrame[(rowIter + 1080 * colBlockIter) + 2073600] =
        varargout_2[rowIter + 1080 * colBlockIter];
    stg2OutFrame[(rowIter + 1080 * colBlockIter) + 4147200] =
        varargout_1[rowIter + 1080 * colBlockIter];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short stg2OutFrame[6220800]
//                unsigned short stg3OutFrame[6220800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel11(
    const unsigned short stg2OutFrame[6220800],
    unsigned short stg3OutFrame[6220800])
{
  unsigned long long threadId;
  int colBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  colBlockIter = static_cast<int>(threadId);
  if (colBlockIter < 6220800) {
    //  GRBG = [(4,1,5)->(1,1), (1,2,3)->(1,2); (3,2,1)->(2,1), (5,1,4)->(2,2)]
    //  Despeckle
    //  Despeckle Algorithm Caller
    stg3OutFrame[colBlockIter] = stg2OutFrame[colBlockIter];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short stg2OutFrame[6220800]
//                unsigned short stg3OutFrame[6220800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel12(
    const unsigned short stg2OutFrame[6220800],
    unsigned short stg3OutFrame[6220800])
{
  unsigned long long threadId;
  int chIter;
  int colIter;
  int rowIter;
  unsigned short winMat[10];
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int>(threadId % 1078ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowIter)) / 1078ULL;
  colIter = static_cast<int>(threadId % 1918ULL);
  threadId = (threadId - static_cast<unsigned long long>(colIter)) / 1918ULL;
  chIter = static_cast<int>(threadId);
  if ((chIter < 3) && (colIter < 1918) && (rowIter < 1078)) {
    winMat[0] = stg2OutFrame[(rowIter + 1080 * colIter) + 2073600 * chIter];
    winMat[1] =
        stg2OutFrame[(rowIter + 1080 * (colIter + 1)) + 2073600 * chIter];
    winMat[2] =
        stg2OutFrame[(rowIter + 1080 * (colIter + 2)) + 2073600 * chIter];
    winMat[3] =
        stg2OutFrame[((rowIter + 1080 * colIter) + 2073600 * chIter) + 1];
    winMat[4] =
        stg2OutFrame[((rowIter + 1080 * (colIter + 1)) + 2073600 * chIter) + 1];
    winMat[5] =
        stg2OutFrame[((rowIter + 1080 * (colIter + 2)) + 2073600 * chIter) + 1];
    winMat[6] =
        stg2OutFrame[((rowIter + 1080 * colIter) + 2073600 * chIter) + 2];
    winMat[7] =
        stg2OutFrame[((rowIter + 1080 * (colIter + 1)) + 2073600 * chIter) + 2];
    winMat[8] =
        stg2OutFrame[((rowIter + 1080 * (colIter + 2)) + 2073600 * chIter) + 2];
    winMat[9] = 0U;
    for (int iter{0}; iter < 5; iter++) {
      int b_colBlockIter;
      b_colBlockIter = 8 - iter;
      for (int rowBlockIter{0}; rowBlockIter < b_colBlockIter; rowBlockIter++) {
        int colBlockIter;
        colBlockIter = (iter + rowBlockIter) + 2;
        if (winMat[iter] > winMat[colBlockIter - 1]) {
          unsigned short t;
          t = winMat[iter];
          winMat[iter] = winMat[colBlockIter - 1];
          winMat[colBlockIter - 1] = t;
        }
      }
    }
    stg3OutFrame[((rowIter + 1080 * (colIter + 1)) + 2073600 * chIter) + 1] =
        winMat[4];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float gainAWB[3]
//                unsigned short stg1OutFrame[2073600]
//                unsigned short stg3OutFrame[6220800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel13(
    const float gainAWB[3], unsigned short stg1OutFrame[2073600],
    unsigned short stg3OutFrame[6220800])
{
  unsigned long long threadId;
  int colIter;
  int rowIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int>(threadId % 1080ULL);
  colIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(rowIter)) / 1080ULL);
  if ((colIter < 1920) && (rowIter < 1080)) {
    float maxVal;
    unsigned short t;
    unsigned short u;
    unsigned short u2;
    maxVal = roundf(static_cast<float>(stg3OutFrame[rowIter + 1080 * colIter]) *
                    gainAWB[0]);
    if (maxVal < 65536.0F) {
      if (maxVal >= 0.0F) {
        t = static_cast<unsigned short>(maxVal);
      } else {
        t = 0U;
      }
    } else if (maxVal >= 65536.0F) {
      t = MAX_uint16_T;
    } else {
      t = 0U;
    }
    if (t >= 4095) {
      u = 4095U;
    } else {
      u = t;
    }
    stg3OutFrame[rowIter + 1080 * colIter] = u;
    maxVal = roundf(
        static_cast<float>(stg3OutFrame[(rowIter + 1080 * colIter) + 2073600]) *
        gainAWB[1]);
    if (maxVal < 65536.0F) {
      if (maxVal >= 0.0F) {
        t = static_cast<unsigned short>(maxVal);
      } else {
        t = 0U;
      }
    } else if (maxVal >= 65536.0F) {
      t = MAX_uint16_T;
    } else {
      t = 0U;
    }
    if (t >= 4095) {
      u2 = 4095U;
    } else {
      u2 = t;
    }
    stg3OutFrame[(rowIter + 1080 * colIter) + 2073600] = u2;
    maxVal = roundf(
        static_cast<float>(stg3OutFrame[(rowIter + 1080 * colIter) + 4147200]) *
        gainAWB[2]);
    if (maxVal < 65536.0F) {
      if (maxVal >= 0.0F) {
        t = static_cast<unsigned short>(maxVal);
      } else {
        t = 0U;
      }
    } else if (maxVal >= 65536.0F) {
      t = MAX_uint16_T;
    } else {
      t = 0U;
    }
    if (t >= 4095) {
      t = 4095U;
    }
    stg3OutFrame[(rowIter + 1080 * colIter) + 4147200] = t;
    u = static_cast<unsigned short>(roundf(
        (0.2126F * static_cast<float>(u) + 0.7152F * static_cast<float>(u2)) +
        0.0722F * static_cast<float>(t)));
    stg1OutFrame[rowIter + 1080 * colIter] = u;
    if (u > 4095) {
      stg1OutFrame[rowIter + 1080 * colIter] = 4095U;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long localHistogram[16384]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel14(
    unsigned long long localHistogram[16384])
{
  unsigned long long threadId;
  int colBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  colBlockIter = static_cast<int>(threadId);
  if (colBlockIter < 16384) {
    //  Histogram Computation
    //  GPU Codegen: Block-wise histogram computation
    //  Histogram per block is stored in the local histogram matrix
    localHistogram[colBlockIter] = 0ULL;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short stg1OutFrame[2073600]
//                unsigned long long localHistogram[16384]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel15(
    const unsigned short stg1OutFrame[2073600],
    unsigned long long localHistogram[16384])
{
  unsigned long long threadId;
  int chIter;
  int colBlockIter;
  int colIter;
  int rowIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int>(threadId % 540ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowIter)) / 540ULL;
  colIter = static_cast<int>(threadId % 960ULL);
  threadId = (threadId - static_cast<unsigned long long>(colIter)) / 960ULL;
  chIter = static_cast<int>(threadId % 2ULL);
  threadId = (threadId - static_cast<unsigned long long>(chIter)) / 2ULL;
  colBlockIter = static_cast<int>(threadId);
  if ((colBlockIter < 2) && (chIter < 2) && (colIter < 960) &&
      (rowIter < 540)) {
    unsigned int qY;
    colBlockIter = colBlockIter * 960 + 1;
    chIter = chIter * 540 + 1;
    qY = stg1OutFrame[((chIter + rowIter) +
                       1080 * ((colBlockIter + colIter) - 1)) -
                      1] +
         1U;
    if (qY > 65535U) {
      qY = 65535U;
    }
    gpu_uint64_atomicAdd(
        &localHistogram
            [((static_cast<int>(qY) +
               ((static_cast<int>((static_cast<double>(chIter) - 1.0) / 540.0 +
                                  1.0) -
                 1)
                << 12)) +
              ((static_cast<int>(
                    (static_cast<double>(colBlockIter) - 1.0) / 960.0 + 1.0) -
                1)
               << 13)) -
             1],
        1ULL);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long globalHistogram[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel16(
    unsigned long long globalHistogram[4096])
{
  unsigned long long threadId;
  int colBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  colBlockIter = static_cast<int>(threadId);
  if (colBlockIter < 4096) {
    //  Local histograms are added to create the final global histogram
    globalHistogram[colBlockIter] = 0ULL;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long localHistogram[16384]
//                unsigned long long globalHistogram[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel17(
    unsigned long long localHistogram[16384],
    unsigned long long globalHistogram[4096])
{
  unsigned long long threadId;
  int chIter;
  int colBlockIter;
  int rowBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  chIter = static_cast<int>(threadId % 4096ULL);
  threadId = (threadId - static_cast<unsigned long long>(chIter)) / 4096ULL;
  rowBlockIter = static_cast<int>(threadId % 2ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowBlockIter)) / 2ULL;
  colBlockIter = static_cast<int>(threadId);
  if ((colBlockIter < 2) && (rowBlockIter < 2) && (chIter < 4096)) {
    gpu_uint64_atomicAdd(
        &globalHistogram[chIter],
        localHistogram[(chIter + (rowBlockIter << 12)) + (colBlockIter << 13)]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned long long globalHistogram[4096]
//                int *bin99Percent
//                int *bin1Percent
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel18(
    const unsigned long long globalHistogram[4096], int *bin99Percent,
    int *bin1Percent)
{
  unsigned long long threadId;
  int iter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  iter = static_cast<int>(threadId);
  if (iter < 4095) {
    threadId = globalHistogram[iter];
    if ((threadId < 207360ULL) && (globalHistogram[iter + 1] >= 207360ULL)) {
      *bin1Percent = iter + 2;
    }
    if ((threadId < 2052864ULL) && (globalHistogram[iter + 1] >= 2052864ULL)) {
      *bin99Percent = iter + 2;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int *bin99Percent
//                const int *bin1Percent
//                unsigned short stg3OutFrame[6220800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel19(
    const int *bin99Percent, const int *bin1Percent,
    unsigned short stg3OutFrame[6220800])
{
  unsigned long long threadId;
  int chIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  chIter = static_cast<int>(threadId);
  if (chIter < 6220800) {
    float maxVal;
    int colBlockIter;
    unsigned short u;
    //  Applying pixel gains
    u = stg3OutFrame[chIter];
    colBlockIter = u - *bin1Percent;
    if (colBlockIter < 0) {
      colBlockIter = 0;
    }
    maxVal = roundf(static_cast<float>((u > *bin1Percent) * colBlockIter) *
                    4095.0F / static_cast<float>(*bin99Percent - *bin1Percent));
    if (maxVal < 65536.0F) {
      if (maxVal >= 0.0F) {
        u = static_cast<unsigned short>(maxVal);
      } else {
        u = 0U;
      }
    } else if (maxVal >= 65536.0F) {
      u = MAX_uint16_T;
    } else {
      u = 0U;
    }
    if (u < 4095) {
      stg3OutFrame[chIter] = u;
    } else {
      stg3OutFrame[chIter] = 4095U;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float rMat[527040]
//                float *b
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void coreVisionPipeline_kernel2(
    const float rMat[527040], float *b)
{
  unsigned long long threadId;
  int indV;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  indV = static_cast<int>(threadId);
  if (indV < 1) {
    *b = rMat[527039];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short stg3OutFrame[6220800]
//                unsigned short stg1OutFrame[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel20(
    const unsigned short stg3OutFrame[6220800],
    unsigned short stg1OutFrame[2073600])
{
  unsigned long long threadId;
  int chIter;
  int colBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  chIter = static_cast<int>(threadId % 1080ULL);
  colBlockIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(chIter)) / 1080ULL);
  if ((colBlockIter < 1920) && (chIter < 1080)) {
    stg1OutFrame[chIter + 1080 * colBlockIter] =
        stg3OutFrame[chIter + 1080 * colBlockIter];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short stg3OutFrame[6220800]
//                unsigned short stg1OutFrame[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel21(
    const unsigned short stg3OutFrame[6220800],
    unsigned short stg1OutFrame[2073600])
{
  unsigned long long threadId;
  int chIter;
  int colBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  chIter = static_cast<int>(threadId % 1080ULL);
  colBlockIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(chIter)) / 1080ULL);
  if ((colBlockIter < 1920) && (chIter < 1080)) {
    stg1OutFrame[chIter + 1080 * colBlockIter] =
        stg3OutFrame[(chIter + 1080 * colBlockIter) + 2073600];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short stg3OutFrame[6220800]
//                unsigned short stg1OutFrame[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel22(
    const unsigned short stg3OutFrame[6220800],
    unsigned short stg1OutFrame[2073600])
{
  unsigned long long threadId;
  int chIter;
  int colBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  chIter = static_cast<int>(threadId % 1080ULL);
  colBlockIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(chIter)) / 1080ULL);
  if ((colBlockIter < 1920) && (chIter < 1080)) {
    stg1OutFrame[chIter + 1080 * colBlockIter] =
        stg3OutFrame[(chIter + 1080 * colBlockIter) + 4147200];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short outImgB[2073600]
//                const unsigned short outImgG[2073600]
//                const unsigned short outImgR[2073600]
//                unsigned short processedFrame[6220800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel23(
    const unsigned short outImgB[2073600],
    const unsigned short outImgG[2073600],
    const unsigned short outImgR[2073600],
    unsigned short processedFrame[6220800])
{
  unsigned long long threadId;
  int chIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  chIter = static_cast<int>(threadId);
  if (chIter < 2073600) {
    processedFrame[3 * (chIter + 1) - 1] = outImgR[chIter];
    processedFrame[3 * (chIter + 1) - 2] = outImgG[chIter];
    processedFrame[3 * (chIter + 1) - 3] = outImgB[chIter];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float gMat[527040]
//                float *b
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void coreVisionPipeline_kernel3(
    const float gMat[527040], float *b)
{
  unsigned long long threadId;
  int indV;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  indV = static_cast<int>(threadId);
  if (indV < 1) {
    *b = gMat[527039];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float bMat[527040]
//                float *b
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void coreVisionPipeline_kernel4(
    const float bMat[527040], float *b)
{
  unsigned long long threadId;
  int indV;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  indV = static_cast<int>(threadId);
  if (indV < 1) {
    *b = bMat[527039];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float gainAWB[3]
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void coreVisionPipeline_kernel5(float gainAWB[3])
{
  unsigned long long threadId;
  int colBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  colBlockIter = static_cast<int>(threadId);
  if (colBlockIter < 3) {
    //  Create dummy kernel to keep the data on GPU
    gainAWB[colBlockIter] = 0.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float meanBChannel
//                const float meanGChannel
//                const float meanRChannel
//                float gainAWB[3]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void coreVisionPipeline_kernel6(
    const float meanBChannel, const float meanGChannel,
    const float meanRChannel, float gainAWB[3])
{
  unsigned long long threadId;
  int bin1Percent;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  bin1Percent = static_cast<int>(threadId);
  if (bin1Percent < 2) {
    float maxVal;
    maxVal = fmaxf(meanRChannel, fmaxf(meanGChannel, meanBChannel));
    gainAWB[0] = maxVal / meanRChannel;
    gainAWB[1] = maxVal / meanGChannel;
    gainAWB[2] = maxVal / meanBChannel;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short outFrameColMajor[2108160]
//                unsigned short inputArray[19200]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel7(
    const unsigned short outFrameColMajor[2108160],
    unsigned short inputArray[19200])
{
  unsigned long long threadId;
  int chIter;
  int colBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  chIter = static_cast<int>(threadId % 10ULL);
  colBlockIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(chIter)) / 10ULL);
  if ((colBlockIter < 1920) && (chIter < 10)) {
    inputArray[chIter + 10 * colBlockIter] =
        outFrameColMajor[chIter + 1098 * colBlockIter];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short inputArray[19200]
//                unsigned int *outputVar
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void coreVisionPipeline_kernel8(
    const unsigned short inputArray[19200], unsigned int *outputVar)
{
  unsigned long long threadId;
  int indV;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  indV = static_cast<int>(threadId);
  if (indV < 1) {
    *outputVar = inputArray[19199];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float meanRChannel
//                const unsigned short outFrameColMajor[2108160]
//                unsigned short stg1OutFrame[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void coreVisionPipeline_kernel9(
    const float meanRChannel, const unsigned short outFrameColMajor[2108160],
    unsigned short stg1OutFrame[2073600])
{
  unsigned long long threadId;
  int colIter;
  int rowIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int>(threadId % 1080ULL);
  colIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(rowIter)) / 1080ULL);
  if ((colIter < 1920) && (rowIter < 1080)) {
    int chIter;
    unsigned int qY;
    chIter = outFrameColMajor[(rowIter + 1098 * colIter) + 18];
    qY = static_cast<unsigned int>(chIter) -
         static_cast<unsigned int>(roundf(meanRChannel));
    if (qY > static_cast<unsigned int>(chIter)) {
      qY = 0U;
    }
    stg1OutFrame[rowIter + 1080 * colIter] = static_cast<unsigned short>(qY);
  }
}

//
// Arguments    : unsigned int in1
//                unsigned int offset
//                unsigned int mask
// Return Type  : unsigned int
//
static __device__ unsigned int shflDown1(unsigned int in1, unsigned int offset,
                                         unsigned int mask)
{
  int *tmp;
  tmp = (int *)&in1;
  *tmp = __shfl_down_sync(mask, *tmp, offset);
  return *(unsigned int *)tmp;
}

//
// Arguments    : float in1
//                unsigned int offset
//                unsigned int mask
// Return Type  : float
//
static __device__ float shflDown1(float in1, unsigned int offset,
                                  unsigned int mask)
{
  int *tmp;
  tmp = (int *)&in1;
  *tmp = __shfl_down_sync(mask, *tmp, offset);
  return *(float *)tmp;
}

//
// Arguments    : unsigned int blockArg
//                unsigned int gridArg
//                const unsigned short input[2073600]
//                unsigned short paddingValue
//                unsigned short output[2073600]
//                unsigned short b_output[2073600]
//                unsigned short c_output[2073600]
//                unsigned short d_output[2073600]
//                unsigned short e_output[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void stencilKernel(
    const unsigned short input[2073600], unsigned short paddingValue,
    unsigned short output[2073600], unsigned short b_output[2073600],
    unsigned short c_output[2073600], unsigned short d_output[2073600],
    unsigned short e_output[2073600])
{
  int workGroupIdTmp;
  int workItemGlobalOutputElemDimIdx;
  int workItemLocalId;
  workItemLocalId = mwGetThreadIndexWithinBlock();
  workGroupIdTmp = mwGetBlockIndex();
  workItemGlobalOutputElemDimIdx =
      workItemLocalId % 16 + ((workGroupIdTmp % 68) << 4);
  workGroupIdTmp /= 68;
  workItemLocalId = workItemLocalId / 16 + (workGroupIdTmp << 4);
  if ((workItemGlobalOutputElemDimIdx < 1080) && (workItemLocalId < 1920)) {
    unsigned int u;
    unsigned short window[9];
    for (int windowIdx{0}; windowIdx < 3; windowIdx++) {
      workGroupIdTmp = (workItemLocalId + windowIdx) - 1;
      for (int b_windowIdx{0}; b_windowIdx < 3; b_windowIdx++) {
        int inputIdx;
        inputIdx = (workItemGlobalOutputElemDimIdx + b_windowIdx) - 1;
        if ((inputIdx >= 0) && (inputIdx < 1080) && (workGroupIdTmp >= 0) &&
            (workGroupIdTmp < 1920)) {
          window[b_windowIdx + 3 * windowIdx] =
              input[inputIdx + 1080 * workGroupIdTmp];
        } else {
          window[b_windowIdx + 3 * windowIdx] = paddingValue;
        }
      }
    }
    //  out1 = [0,0,0;0,1,0;0,0,0] // Center
    //  out2 = [0,1,0;1,0,1;0,1,0] // LRTB
    //  out3 = [1,0,1;0,0,0;1,0,1] // Corner
    //  out4 = [0,0,0;1,0,1;0,0,0] // LR
    //  out5 = [0,1,0;0,0,0;0,1,0] // TB
    //  Stencil function
    output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] = window[4];
    u = static_cast<unsigned int>(window[1]) + window[3];
    if (u > 65535U) {
      u = 65535U;
    }
    u += window[7];
    if (u > 65535U) {
      u = 65535U;
    }
    u += window[5];
    if (u > 65535U) {
      u = 65535U;
    }
    b_output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] =
        static_cast<unsigned short>(roundf(static_cast<float>(u) * 0.25F));
    u = static_cast<unsigned int>(window[0]) + window[6];
    if (u > 65535U) {
      u = 65535U;
    }
    u += window[2];
    if (u > 65535U) {
      u = 65535U;
    }
    u += window[8];
    if (u > 65535U) {
      u = 65535U;
    }
    c_output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] =
        static_cast<unsigned short>(roundf(static_cast<float>(u) * 0.25F));
    u = static_cast<unsigned int>(window[1]) + window[7];
    if (u > 65535U) {
      u = 65535U;
    }
    d_output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] =
        static_cast<unsigned short>(roundf(static_cast<float>(u) * 0.5F));
    u = static_cast<unsigned int>(window[3]) + window[5];
    if (u > 65535U) {
      u = 65535U;
    }
    e_output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] =
        static_cast<unsigned short>(roundf(static_cast<float>(u) * 0.5F));
  }
}

//
// Arguments    : unsigned int val
//                unsigned int lane
//                unsigned int mask
// Return Type  : unsigned int
//
static __device__ unsigned int
threadGroupReduction(unsigned int val, unsigned int lane, unsigned int mask)
{
  unsigned int activeSize;
  unsigned int offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    unsigned int other;
    other = shflDown1(val, offset, mask);
    if (lane + offset < activeSize) {
      activeSize = val;
      //  Helper function
      val += other;
      if (val < activeSize) {
        val = MAX_uint32_T;
      }
    }
    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }
  return val;
}

//
// Arguments    : float val
//                unsigned int lane
//                unsigned int mask
// Return Type  : float
//
static __device__ float threadGroupReduction(float val, unsigned int lane,
                                             unsigned int mask)
{
  unsigned int activeSize;
  unsigned int offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    float other;
    other = shflDown1(val, offset, mask);
    if (lane + offset < activeSize) {
      //  Helper function
      val += other;
    }
    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }
  return val;
}

//
// Arguments    : float val
//                unsigned int mask
//                unsigned int numActiveWarps
// Return Type  : float
//
static __device__ float workGroupReduction(float val, unsigned int mask,
                                           unsigned int numActiveWarps)
{
  __shared__ float shared[32];
  unsigned int lane;
  unsigned int threadId;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  threadId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  lane = threadId % warpSize;
  threadId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[threadId] = val;
  }
  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (threadId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }
  return val;
}

//
// Arguments    : unsigned int val
//                unsigned int mask
//                unsigned int numActiveWarps
// Return Type  : unsigned int
//
static __device__ unsigned int workGroupReduction(unsigned int val,
                                                  unsigned int mask,
                                                  unsigned int numActiveWarps)
{
  __shared__ unsigned int shared[32];
  unsigned int lane;
  unsigned int threadId;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  threadId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  lane = threadId % warpSize;
  threadId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[threadId] = val;
  }
  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (threadId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }
  return val;
}

//
// Arguments    : const unsigned short inputFrame[2108160]
//                float gainAWB[3]
//                double runAWB
//                unsigned short processedFrame[6220800]
// Return Type  : void
//
void coreVisionPipeline(const unsigned short inputFrame[2108160],
                        float gainAWB[3], double runAWB,
                        unsigned short processedFrame[6220800])
{
  unsigned long long(*gpu_localHistogram)[16384];
  unsigned long long(*gpu_globalHistogram)[4096];
  float(*gpu_bMat)[527040];
  float(*gpu_gMat)[527040];
  float(*gpu_rMat)[527040];
  float b;
  float c;
  float d;
  float *b_gpu_tmp;
  float *c_gpu_tmp;
  float *gpu_tmp;
  int bin1Percent;
  int bin99Percent;
  unsigned int outputVar;
  int *gpu_bin1Percent;
  int *gpu_bin99Percent;
  unsigned int *gpu_outputVar;
  unsigned short(*gpu_stg2OutFrame)[6220800];
  unsigned short(*gpu_stg3OutFrame)[6220800];
  unsigned short(*gpu_outFrameColMajor)[2108160];
  unsigned short(*gpu_outImgB)[2073600];
  unsigned short(*gpu_outImgG)[2073600];
  unsigned short(*gpu_outImgR)[2073600];
  unsigned short(*gpu_stg1OutFrame)[2073600];
  unsigned short(*gpu_varargout_1)[2073600];
  unsigned short(*gpu_varargout_2)[2073600];
  unsigned short(*gpu_varargout_3)[2073600];
  unsigned short(*gpu_varargout_4)[2073600];
  unsigned short(*gpu_varargout_5)[2073600];
  unsigned short(*gpu_inputArray)[19200];
  if (!isInitialized_coreVisionPipeline) {
    coreVisionPipeline_initialize();
  }
  mwCudaMalloc(&gpu_outImgB, 4147200ULL);
  mwCudaMalloc(&gpu_outImgG, 4147200ULL);
  mwCudaMalloc(&gpu_outImgR, 4147200ULL);
  mwCudaMalloc(&gpu_bin1Percent, 4ULL);
  mwCudaMalloc(&gpu_bin99Percent, 4ULL);
  mwCudaMalloc(&gpu_globalHistogram, 32768ULL);
  mwCudaMalloc(&gpu_localHistogram, 131072ULL);
  mwCudaMalloc(&gpu_stg3OutFrame, 12441600ULL);
  mwCudaMalloc(&gpu_stg2OutFrame, 12441600ULL);
  mwCudaMalloc(&gpu_varargout_5, 4147200ULL);
  mwCudaMalloc(&gpu_varargout_4, 4147200ULL);
  mwCudaMalloc(&gpu_varargout_3, 4147200ULL);
  mwCudaMalloc(&gpu_varargout_2, 4147200ULL);
  mwCudaMalloc(&gpu_varargout_1, 4147200ULL);
  mwCudaMalloc(&gpu_stg1OutFrame, 4147200ULL);
  mwCudaMalloc(&c_gpu_tmp, 4ULL);
  mwCudaMalloc(&gpu_outputVar, 4ULL);
  mwCudaMalloc(&gpu_inputArray, 38400ULL);
  mwCudaMalloc(&b_gpu_tmp, 4ULL);
  mwCudaMalloc(&gpu_tmp, 4ULL);
  mwCudaMalloc(&gpu_outFrameColMajor, 4216320ULL);
  mwCudaMalloc(&gpu_rMat, 2108160ULL);
  mwCudaMalloc(&gpu_bMat, 2108160ULL);
  mwCudaMalloc(&gpu_gMat, 2108160ULL);
  //  AWB Gain update
  if (runAWB != 0.0) {
    //  Compute the mean of RGB channels in the bayer patch
    coreVisionPipeline_kernel1<<<dim3(259U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        inputFrame, *gpu_gMat, *gpu_bMat, *gpu_rMat);
    coreVisionPipeline_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_rMat, gpu_tmp);
    coder_reduce0<<<dim3(1030U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_rMat,
                                                               gpu_tmp);
    hipMemcpy(&b, gpu_tmp, 4ULL, hipMemcpyDeviceToHost);
    coreVisionPipeline_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_gMat, b_gpu_tmp);
    coder_reduce1<<<dim3(1030U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_gMat,
                                                               b_gpu_tmp);
    hipMemcpy(&c, b_gpu_tmp, 4ULL, hipMemcpyDeviceToHost);
    coreVisionPipeline_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_bMat, c_gpu_tmp);
    coder_reduce2<<<dim3(1030U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_bMat,
                                                               c_gpu_tmp);
    hipMemcpy(&d, c_gpu_tmp, 4ULL, hipMemcpyDeviceToHost);
    //  Create dummy kernel to keep the data on GPU
    coreVisionPipeline_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        gainAWB);
    coreVisionPipeline_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        d / 2.10816E+6F, 1.2F * c / 2.10816E+6F, b / 2.10816E+6F, gainAWB);
  }
  //  Process input frame
  //  Input Frame row-major to column-major conversion
  transposeImpl((unsigned short *)&inputFrame[0], &(*gpu_outFrameColMajor)[0],
                1920, 1098, false);
  //  outFrameColMajor = inputFrame;
  //  Split Frame and Black Correction
  //  Constants:
  //  Rows for black correction cropping
  //  Rows for black level estimation
  //  Input Size
  //  Estimating the black value in both the frames
  coreVisionPipeline_kernel7<<<dim3(38U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_outFrameColMajor, *gpu_inputArray);
  coreVisionPipeline_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_inputArray, gpu_outputVar);
  coder_reduce3<<<dim3(38U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_inputArray,
                                                           gpu_outputVar);
  hipMemcpy(&outputVar, gpu_outputVar, 4ULL, hipMemcpyDeviceToHost);
  if (outputVar > 65535U) {
    outputVar = 65535U;
  }
  //  Subtract mean black and remove the crop the blackCorrection rows
  coreVisionPipeline_kernel9<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      static_cast<float>(outputVar) / 19200.0F, *gpu_outFrameColMajor,
      *gpu_stg1OutFrame);
  //  Debayer
  //  Function to call debayer algorithm on input image.
  //     %% Sizes and constants
  //  Debayer Function
  //  Convolve image with masks
  stencilKernel<<<8160U, 256U>>>(*gpu_stg1OutFrame, 0U, *gpu_varargout_1,
                                 *gpu_varargout_2, *gpu_varargout_3,
                                 *gpu_varargout_4, *gpu_varargout_5);
  //  Plane Ordering
  //  Since stencil kernel performs
  //  BayerFormat = [(R-plane,G-plane,B-plane)->(Row coord, Col coord)]
  //  Following the RGGB bayer format
  coreVisionPipeline_kernel10<<<dim3(1013U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_varargout_4, *gpu_varargout_5, *gpu_varargout_3, *gpu_varargout_2,
      *gpu_varargout_1, *gpu_stg2OutFrame);
  //  GRBG = [(4,1,5)->(1,1), (1,2,3)->(1,2); (3,2,1)->(2,1), (5,1,4)->(2,2)]
  //  Despeckle
  //  Despeckle Algorithm Caller
  coreVisionPipeline_kernel11<<<dim3(12150U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_stg2OutFrame, *gpu_stg3OutFrame);
  coreVisionPipeline_kernel12<<<dim3(12115U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_stg2OutFrame, *gpu_stg3OutFrame);
  //  White Balance
  //  Luma Gain
  //  GPU Pragmas
  //  Compute sizes and declaring constants
  //  Compute Luminance
  coreVisionPipeline_kernel13<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      gainAWB, *gpu_stg1OutFrame, *gpu_stg3OutFrame);
  //  Histogram Computation
  //  GPU Codegen: Block-wise histogram computation
  //  Histogram per block is stored in the local histogram matrix
  coreVisionPipeline_kernel14<<<dim3(32U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_localHistogram);
  coreVisionPipeline_kernel15<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_stg1OutFrame, *gpu_localHistogram);
  //  Local histograms are added to create the final global histogram
  coreVisionPipeline_kernel16<<<dim3(8U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_globalHistogram);
  coreVisionPipeline_kernel17<<<dim3(32U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_localHistogram, *gpu_globalHistogram);
  //  Histogram equalization
  //  Cumulative histogram values
  callThrustScan1D(&(*gpu_globalHistogram)[0], false, 4096);
  //  Identify 90-th percentile bin and computing the smoothing factor
  bin1Percent = 0;
  bin99Percent = 0;
  hipMemcpy(gpu_bin99Percent, &bin99Percent, 4ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_bin1Percent, &bin1Percent, 4ULL, hipMemcpyHostToDevice);
  coreVisionPipeline_kernel18<<<dim3(8U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_globalHistogram, gpu_bin99Percent, gpu_bin1Percent);
  //  Applying pixel gains
  coreVisionPipeline_kernel19<<<dim3(12150U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      gpu_bin99Percent, gpu_bin1Percent, *gpu_stg3OutFrame);
  //  Planar to packed
  //  Assuming input planar array pointer is row-major of size RowsxColsx3
  coreVisionPipeline_kernel20<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_stg3OutFrame, *gpu_stg1OutFrame);
  transposeImpl(&(*gpu_stg1OutFrame)[0], &(*gpu_outImgR)[0], 1080, 1920, false);
  coreVisionPipeline_kernel21<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_stg3OutFrame, *gpu_stg1OutFrame);
  transposeImpl(&(*gpu_stg1OutFrame)[0], &(*gpu_outImgG)[0], 1080, 1920, false);
  coreVisionPipeline_kernel22<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_stg3OutFrame, *gpu_stg1OutFrame);
  transposeImpl(&(*gpu_stg1OutFrame)[0], &(*gpu_outImgB)[0], 1080, 1920, false);
  coreVisionPipeline_kernel23<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_outImgB, *gpu_outImgG, *gpu_outImgR, processedFrame);
  //      processedFrame = outFrameRGB;
  mwCudaFree(&(*gpu_gMat)[0]);
  mwCudaFree(&(*gpu_bMat)[0]);
  mwCudaFree(&(*gpu_rMat)[0]);
  mwCudaFree(&(*gpu_outFrameColMajor)[0]);
  mwCudaFree(gpu_tmp);
  mwCudaFree(b_gpu_tmp);
  mwCudaFree(&(*gpu_inputArray)[0]);
  mwCudaFree(gpu_outputVar);
  mwCudaFree(c_gpu_tmp);
  mwCudaFree(&(*gpu_stg1OutFrame)[0]);
  mwCudaFree(&(*gpu_varargout_1)[0]);
  mwCudaFree(&(*gpu_varargout_2)[0]);
  mwCudaFree(&(*gpu_varargout_3)[0]);
  mwCudaFree(&(*gpu_varargout_4)[0]);
  mwCudaFree(&(*gpu_varargout_5)[0]);
  mwCudaFree(&(*gpu_stg2OutFrame)[0]);
  mwCudaFree(&(*gpu_stg3OutFrame)[0]);
  mwCudaFree(&(*gpu_localHistogram)[0]);
  mwCudaFree(&(*gpu_globalHistogram)[0]);
  mwCudaFree(gpu_bin99Percent);
  mwCudaFree(gpu_bin1Percent);
  mwCudaFree(&(*gpu_outImgR)[0]);
  mwCudaFree(&(*gpu_outImgG)[0]);
  mwCudaFree(&(*gpu_outImgB)[0]);
}

//
// File trailer for coreVisionPipeline.cu
//
// [EOF]
//
