//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: isFrameSaturated_initialize.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 13-Mar-2023 12:15:40
//

// Include Files
#include "isFrameSaturated_initialize.h"
#include "isFrameSaturated_data.h"
#include "MWMemoryManager.hpp"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void isFrameSaturated_initialize()
{
  mwMemoryManagerInit(256U, 0U, 8U, 2048U);
  hipGetLastError();
  isInitialized_isFrameSaturated = true;
}

//
// File trailer for isFrameSaturated_initialize.cu
//
// [EOF]
//
