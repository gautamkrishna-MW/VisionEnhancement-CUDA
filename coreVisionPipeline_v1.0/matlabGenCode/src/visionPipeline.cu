#include "hip/hip_runtime.h"
//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: visionPipeline.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 13-Feb-2023 14:42:18
//

// Include Files
#include "visionPipeline.h"
#include "visionPipeline_data.h"
#include "visionPipeline_initialize.h"
#include "MWAtomicUtility.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWScanFunctors.h"
#include "MWScanUtility.h"
#include "MWShuffleUtility.h"

// Function Declarations
static __device__ float atomicOpreal32_T(float *address, float value);

static __global__ void coder_reduce0(const float inputVar[129600],
                                     float *outputVar);

static __global__ void coder_reduce1(const float inputVar[129600],
                                     float *outputVar);

static __global__ void coder_reduce2(const float inputVar[129600],
                                     float *outputVar);

static __device__ int shflDown1(int in1, unsigned int offset,
                                unsigned int mask);

static __device__ float shflDown1(float in1, unsigned int offset,
                                  unsigned int mask);

static __global__ void stencilKernel(const unsigned short input[2073600],
                                     unsigned short paddingValue,
                                     unsigned short output[2073600]);

static __global__ void stencilKernel(const unsigned short input[2073600],
                                     unsigned short paddingValue,
                                     unsigned short output[2073600],
                                     unsigned short b_output[2073600],
                                     unsigned short c_output[2073600],
                                     unsigned short d_output[2073600],
                                     unsigned short e_output[2073600]);

static __device__ int threadGroupReduction(int val, unsigned int lane,
                                           unsigned int mask);

static __device__ float threadGroupReduction(float val, unsigned int lane,
                                             unsigned int mask);

static __global__ void
visionPipeline_kernel1(const unsigned short inputFrame[2108160], int *y);

static __global__ void visionPipeline_kernel10(const float gMat[129600],
                                               float *b);

static __global__ void visionPipeline_kernel11(const float bMat[129600],
                                               float *b);

static __global__ void
visionPipeline_kernel12(const float meanBChannel, const float meanGChannel,
                        const float meanRChannel, float *gainBChannel,
                        float *gainGChannel, float *gainRChannel);

static __global__ void
visionPipeline_kernel13(const float *gainBChannel, const float *gainGChannel,
                        const float *gainRChannel, double x[2073600],
                        unsigned short stg3OutFrame[6220800]);

static __global__ void
visionPipeline_kernel14(unsigned short frameLuma[2073600], double x[2073600]);

static __global__ void
visionPipeline_kernel15(unsigned long long localHistogram[16384]);

static __global__ void
visionPipeline_kernel16(const unsigned short frameLuma[2073600],
                        unsigned long long localHistogram[16384]);

static __global__ void
visionPipeline_kernel17(unsigned long long globalHistogram[4096]);

static __global__ void
visionPipeline_kernel18(unsigned long long localHistogram[16384],
                        unsigned long long globalHistogram[4096]);

static __global__ void
visionPipeline_kernel19(const unsigned long long globalHistogram[4096],
                        int *bin1Percent, int *bin99Percent, int *binVal);

static __global__ void
visionPipeline_kernel2(const unsigned short inputFrame[2108160], int *y);

static __global__ void visionPipeline_kernel20(const int *bin99Percent,
                                               const int *bin1Percent,
                                               double pixelGainLUT[4096]);

static __global__ void
visionPipeline_kernel21(const double pixelGainLUT[4096],
                        const unsigned short stg3OutFrame[6220800],
                        unsigned short outFrame[6220800]);

static __global__ void
visionPipeline_kernel3(const double meanBlackValue,
                       const unsigned short inputFrame[2108160],
                       unsigned short frameLuma[2073600]);

static __global__ void
visionPipeline_kernel4(const unsigned short varargout_5[2073600],
                       const unsigned short varargout_4[2073600],
                       const unsigned short varargout_3[2073600],
                       const unsigned short varargout_2[2073600],
                       const unsigned short varargout_1[2073600],
                       unsigned short stg2OutFrame[6220800]);

static __global__ void
visionPipeline_kernel5(const unsigned short stg2OutFrame[6220800],
                       unsigned short frameLuma[2073600]);

static __global__ void
visionPipeline_kernel6(const unsigned short stg2OutFrame[6220800],
                       unsigned short frameLuma[2073600]);

static __global__ void
visionPipeline_kernel7(const unsigned short stg2OutFrame[6220800],
                       unsigned short frameLuma[2073600]);

static __global__ void
visionPipeline_kernel8(const unsigned short whitePatch[518400],
                       float gMat[129600], float bMat[129600],
                       float rMat[129600]);

static __global__ void visionPipeline_kernel9(const float rMat[129600],
                                              float *b);

static __device__ int workGroupReduction(int val, unsigned int mask,
                                         unsigned int numActiveWarps);

static __device__ float workGroupReduction(float val, unsigned int mask,
                                           unsigned int numActiveWarps);

// Function Definitions
//
// Arguments    : float *address
//                float value
// Return Type  : float
//
static __device__ float atomicOpreal32_T(float *address, float value)
{
  unsigned int old;
  unsigned int *address_as_up;
  address_as_up = (unsigned int *)address;
  old = *address_as_up;
  float input2;
  unsigned int assumed;
  do {
    assumed = old;
    input2 = __uint_as_float(old);
    old = atomicCAS(address_as_up, old, __float_as_uint(value + input2));
  } while (assumed != old);
  return __uint_as_float(old);
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float inputVar[129600]
//                float *outputVar
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void coder_reduce0(const float inputVar[129600],
                                                  float *outputVar)
{
  float input1;
  unsigned int blockStride;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  if (static_cast<unsigned int>(mwGetBlockIndex()) == 129599U / blockStride) {
    m = 129599U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 129598U) {
    input1 = inputVar[threadId];
  }
  m = __ballot_sync(MAX_uint32_T, threadId <= 129598U);
  for (unsigned int idx{threadId + threadStride}; idx <= 129598U;
       idx += threadStride) {
    float input2;
    input2 = inputVar[idx];
    input1 += input2;
  }
  input1 = workGroupReduction(input1, m, blockStride);
  if ((threadId <= 129598U) && (thBlkId == 0U)) {
    atomicOpreal32_T(&outputVar[0], input1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float inputVar[129600]
//                float *outputVar
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void coder_reduce1(const float inputVar[129600],
                                                  float *outputVar)
{
  float input1;
  unsigned int blockStride;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  if (static_cast<unsigned int>(mwGetBlockIndex()) == 129599U / blockStride) {
    m = 129599U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 129598U) {
    input1 = inputVar[threadId];
  }
  m = __ballot_sync(MAX_uint32_T, threadId <= 129598U);
  for (unsigned int idx{threadId + threadStride}; idx <= 129598U;
       idx += threadStride) {
    float input2;
    input2 = inputVar[idx];
    input1 += input2;
  }
  input1 = workGroupReduction(input1, m, blockStride);
  if ((threadId <= 129598U) && (thBlkId == 0U)) {
    atomicOpreal32_T(&outputVar[0], input1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float inputVar[129600]
//                float *outputVar
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void coder_reduce2(const float inputVar[129600],
                                                  float *outputVar)
{
  float input1;
  unsigned int blockStride;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  if (static_cast<unsigned int>(mwGetBlockIndex()) == 129599U / blockStride) {
    m = 129599U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 129598U) {
    input1 = inputVar[threadId];
  }
  m = __ballot_sync(MAX_uint32_T, threadId <= 129598U);
  for (unsigned int idx{threadId + threadStride}; idx <= 129598U;
       idx += threadStride) {
    float input2;
    input2 = inputVar[idx];
    input1 += input2;
  }
  input1 = workGroupReduction(input1, m, blockStride);
  if ((threadId <= 129598U) && (thBlkId == 0U)) {
    atomicOpreal32_T(&outputVar[0], input1);
  }
}

//
// Arguments    : int in1
//                unsigned int offset
//                unsigned int mask
// Return Type  : int
//
static __device__ int shflDown1(int in1, unsigned int offset, unsigned int mask)
{
  in1 = __shfl_down_sync(mask, in1, offset);
  return in1;
}

//
// Arguments    : float in1
//                unsigned int offset
//                unsigned int mask
// Return Type  : float
//
static __device__ float shflDown1(float in1, unsigned int offset,
                                  unsigned int mask)
{
  int *tmp;
  tmp = (int *)&in1;
  *tmp = __shfl_down_sync(mask, *tmp, offset);
  return *(float *)tmp;
}

//
// Arguments    : unsigned int blockArg
//                unsigned int gridArg
//                const unsigned short input[2073600]
//                unsigned short paddingValue
//                unsigned short output[2073600]
//                unsigned short b_output[2073600]
//                unsigned short c_output[2073600]
//                unsigned short d_output[2073600]
//                unsigned short e_output[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void stencilKernel(
    const unsigned short input[2073600], unsigned short paddingValue,
    unsigned short output[2073600], unsigned short b_output[2073600],
    unsigned short c_output[2073600], unsigned short d_output[2073600],
    unsigned short e_output[2073600])
{
  int workGroupIdTmp;
  int workItemGlobalOutputElemDimIdx;
  int workItemLocalId;
  workItemLocalId = mwGetThreadIndexWithinBlock();
  workGroupIdTmp = mwGetBlockIndex();
  workItemGlobalOutputElemDimIdx =
      workItemLocalId % 16 + ((workGroupIdTmp % 68) << 4);
  workGroupIdTmp /= 68;
  workItemLocalId = workItemLocalId / 16 + (workGroupIdTmp << 4);
  if ((workItemGlobalOutputElemDimIdx < 1080) && (workItemLocalId < 1920)) {
    unsigned int u;
    unsigned short window[9];
    for (int windowIdx{0}; windowIdx < 3; windowIdx++) {
      workGroupIdTmp = (workItemLocalId + windowIdx) - 1;
      for (int b_windowIdx{0}; b_windowIdx < 3; b_windowIdx++) {
        int inputIdx;
        inputIdx = (workItemGlobalOutputElemDimIdx + b_windowIdx) - 1;
        if ((inputIdx >= 0) && (inputIdx < 1080) && (workGroupIdTmp >= 0) &&
            (workGroupIdTmp < 1920)) {
          window[b_windowIdx + 3 * windowIdx] =
              input[inputIdx + 1080 * workGroupIdTmp];
        } else {
          window[b_windowIdx + 3 * windowIdx] = paddingValue;
        }
      }
    }
    //  out1 = [0,0,0;0,1,0;0,0,0] // Center
    //  out2 = [0,1,0;1,0,1;0,1,0] // LRTB
    //  out3 = [1,0,1;0,0,0;1,0,1] // Corner
    //  out4 = [0,0,0;1,0,1;0,0,0] // LR
    //  out5 = [0,1,0;0,0,0;0,1,0] // TB
    //  Stencil function
    output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] = window[4];
    u = static_cast<unsigned int>(window[1]) + window[3];
    if (u > 65535U) {
      u = 65535U;
    }
    u += window[7];
    if (u > 65535U) {
      u = 65535U;
    }
    u += window[5];
    if (u > 65535U) {
      u = 65535U;
    }
    b_output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] =
        static_cast<unsigned short>(roundf(static_cast<float>(u) * 0.25F));
    u = static_cast<unsigned int>(window[0]) + window[6];
    if (u > 65535U) {
      u = 65535U;
    }
    u += window[2];
    if (u > 65535U) {
      u = 65535U;
    }
    u += window[8];
    if (u > 65535U) {
      u = 65535U;
    }
    c_output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] =
        static_cast<unsigned short>(roundf(static_cast<float>(u) * 0.25F));
    u = static_cast<unsigned int>(window[1]) + window[7];
    if (u > 65535U) {
      u = 65535U;
    }
    d_output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] =
        static_cast<unsigned short>(roundf(static_cast<float>(u) * 0.5F));
    u = static_cast<unsigned int>(window[3]) + window[5];
    if (u > 65535U) {
      u = 65535U;
    }
    e_output[workItemGlobalOutputElemDimIdx + 1080 * workItemLocalId] =
        static_cast<unsigned short>(roundf(static_cast<float>(u) * 0.5F));
  }
}

//
// Arguments    : unsigned int blockArg
//                unsigned int gridArg
//                const unsigned short input[2073600]
//                unsigned short paddingValue
//                unsigned short output[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void stencilKernel(
    const unsigned short input[2073600], unsigned short paddingValue,
    unsigned short output[2073600])
{
  int b_workItemGlobalOutputElemDimIdx;
  int workGroupIdTmp;
  int workItemGlobalOutputElemDimIdx;
  int workItemLocalId;
  workItemLocalId = mwGetThreadIndexWithinBlock();
  workGroupIdTmp = mwGetBlockIndex();
  workItemGlobalOutputElemDimIdx =
      workItemLocalId % 16 + ((workGroupIdTmp % 68) << 4);
  workGroupIdTmp /= 68;
  b_workItemGlobalOutputElemDimIdx =
      workItemLocalId / 16 + (workGroupIdTmp << 4);
  if ((workItemGlobalOutputElemDimIdx < 1080) &&
      (b_workItemGlobalOutputElemDimIdx < 1920)) {
    unsigned short window[9];
    for (int windowIdx{0}; windowIdx < 3; windowIdx++) {
      workItemLocalId = (b_workItemGlobalOutputElemDimIdx + windowIdx) - 1;
      for (int b_windowIdx{0}; b_windowIdx < 3; b_windowIdx++) {
        workGroupIdTmp = (workItemGlobalOutputElemDimIdx + b_windowIdx) - 1;
        if ((workGroupIdTmp >= 0) && (workGroupIdTmp < 1080) &&
            (workItemLocalId >= 0) && (workItemLocalId < 1920)) {
          window[b_windowIdx + 3 * windowIdx] =
              input[workGroupIdTmp + 1080 * workItemLocalId];
        } else {
          window[b_windowIdx + 3 * windowIdx] = paddingValue;
        }
      }
    }
    //  Median filter stencil kernel implementation.
    //  Apply median filter only when the mid-value is beyond sensitivity level.
    //  Sort values and replace mid-value with median
    for (int windowIdx{0}; windowIdx < 9; windowIdx++) {
      workItemLocalId = 7 - windowIdx;
      for (int b_windowIdx{0}; b_windowIdx <= workItemLocalId; b_windowIdx++) {
        workGroupIdTmp = (windowIdx + b_windowIdx) + 1;
        if (window[workGroupIdTmp] < window[windowIdx]) {
          unsigned short u;
          u = window[workGroupIdTmp];
          window[workGroupIdTmp] = window[windowIdx];
          //  Function to swap values
          window[windowIdx] = u;
        }
      }
    }
    output[workItemGlobalOutputElemDimIdx +
           1080 * b_workItemGlobalOutputElemDimIdx] = window[4];
  }
}

//
// Arguments    : int val
//                unsigned int lane
//                unsigned int mask
// Return Type  : int
//
static __device__ int threadGroupReduction(int val, unsigned int lane,
                                           unsigned int mask)
{
  unsigned int activeSize;
  unsigned int offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    int other;
    other = shflDown1(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }
    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }
  return val;
}

//
// Arguments    : float val
//                unsigned int lane
//                unsigned int mask
// Return Type  : float
//
static __device__ float threadGroupReduction(float val, unsigned int lane,
                                             unsigned int mask)
{
  unsigned int activeSize;
  unsigned int offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    float other;
    other = shflDown1(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }
    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }
  return val;
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short inputFrame[2108160]
//                int *y
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void visionPipeline_kernel1(
    const unsigned short inputFrame[2108160], int *y)
{
  unsigned long long threadId;
  int tmpIdx;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    //  Split Frame and Black Correction
    //  Constants:
    //  Rows for black correction cropping
    //  Rows for black level estimation
    //  Estimating the black value in both the frames
    *y = inputFrame[0];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float gMat[129600]
//                float *b
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void visionPipeline_kernel10(
    const float gMat[129600], float *b)
{
  unsigned long long threadId;
  int indV;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  indV = static_cast<int>(threadId);
  if (indV < 1) {
    *b = gMat[129599];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float bMat[129600]
//                float *b
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void visionPipeline_kernel11(
    const float bMat[129600], float *b)
{
  unsigned long long threadId;
  int indV;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  indV = static_cast<int>(threadId);
  if (indV < 1) {
    *b = bMat[129599];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float meanBChannel
//                const float meanGChannel
//                const float meanRChannel
//                float *gainBChannel
//                float *gainGChannel
//                float *gainRChannel
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void visionPipeline_kernel12(
    const float meanBChannel, const float meanGChannel,
    const float meanRChannel, float *gainBChannel, float *gainGChannel,
    float *gainRChannel)
{
  unsigned long long threadId;
  int y;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  y = static_cast<int>(threadId);
  if (y < 2) {
    float pixVal;
    //  Create dummy kernel to keep the data on GPU
    pixVal = fmaxf(meanRChannel, fmaxf(meanGChannel, meanBChannel));
    *gainRChannel = pixVal / meanRChannel;
    *gainGChannel = pixVal / meanGChannel;
    *gainBChannel = pixVal / meanBChannel;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float *gainBChannel
//                const float *gainGChannel
//                const float *gainRChannel
//                double x[2073600]
//                unsigned short stg3OutFrame[6220800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel13(
    const float *gainBChannel, const float *gainGChannel,
    const float *gainRChannel, double x[2073600],
    unsigned short stg3OutFrame[6220800])
{
  unsigned long long threadId;
  int colIter;
  int rowIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int>(threadId % 1080ULL);
  colIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(rowIter)) / 1080ULL);
  if ((colIter < 1920) && (rowIter < 1080)) {
    float pixVal;
    unsigned short u;
    unsigned short u2;
    unsigned short u3;
    pixVal = static_cast<float>(stg3OutFrame[rowIter + 1080 * colIter]) *
             *gainRChannel;
    pixVal = static_cast<float>(pixVal >= 4095.0F) * 4095.0F +
             static_cast<float>(pixVal < 4095.0F) * pixVal;
    pixVal = roundf(pixVal);
    if (pixVal < 65536.0F) {
      if (pixVal >= 0.0F) {
        u = static_cast<unsigned short>(pixVal);
      } else {
        u = 0U;
      }
    } else if (pixVal >= 65536.0F) {
      u = MAX_uint16_T;
    } else {
      u = 0U;
    }
    stg3OutFrame[rowIter + 1080 * colIter] = u;
    pixVal =
        static_cast<float>(stg3OutFrame[(rowIter + 1080 * colIter) + 2073600]) *
        *gainGChannel;
    pixVal = static_cast<float>(pixVal >= 4095.0F) * 4095.0F +
             static_cast<float>(pixVal < 4095.0F) * pixVal;
    pixVal = roundf(pixVal);
    if (pixVal < 65536.0F) {
      if (pixVal >= 0.0F) {
        u2 = static_cast<unsigned short>(pixVal);
      } else {
        u2 = 0U;
      }
    } else if (pixVal >= 65536.0F) {
      u2 = MAX_uint16_T;
    } else {
      u2 = 0U;
    }
    stg3OutFrame[(rowIter + 1080 * colIter) + 2073600] = u2;
    pixVal =
        static_cast<float>(stg3OutFrame[(rowIter + 1080 * colIter) + 4147200]) *
        *gainBChannel;
    pixVal = static_cast<float>(pixVal >= 4095.0F) * 4095.0F +
             static_cast<float>(pixVal < 4095.0F) * pixVal;
    pixVal = roundf(pixVal);
    if (pixVal < 65536.0F) {
      if (pixVal >= 0.0F) {
        u3 = static_cast<unsigned short>(pixVal);
      } else {
        u3 = 0U;
      }
    } else if (pixVal >= 65536.0F) {
      u3 = MAX_uint16_T;
    } else {
      u3 = 0U;
    }
    stg3OutFrame[(rowIter + 1080 * colIter) + 4147200] = u3;
    x[rowIter + 1080 * colIter] =
        (0.2126 * static_cast<double>(u) + 0.7152 * static_cast<double>(u2)) +
        0.0722 * static_cast<double>(u3);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned short frameLuma[2073600]
//                double x[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel14(
    unsigned short frameLuma[2073600], double x[2073600])
{
  unsigned long long threadId;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId);
  if (k < 2073600) {
    int rowBlockIter;
    rowBlockIter = static_cast<int>(round(x[k]));
    x[k] = rowBlockIter;
    frameLuma[k] = static_cast<unsigned short>(rowBlockIter);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long localHistogram[16384]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel15(
    unsigned long long localHistogram[16384])
{
  unsigned long long threadId;
  int rowBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowBlockIter = static_cast<int>(threadId);
  if (rowBlockIter < 16384) {
    //  Histogram Computation
    //  GPU Codegen: Block-wise histogram computation
    //  Histogram per block is stored in the local histogram matrix
    localHistogram[rowBlockIter] = 0ULL;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short frameLuma[2073600]
//                unsigned long long localHistogram[16384]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel16(
    const unsigned short frameLuma[2073600],
    unsigned long long localHistogram[16384])
{
  unsigned long long threadId;
  int colIter;
  int rowBlockIter;
  int rowIter;
  int tmpVal;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowIter = static_cast<int>(threadId % 540ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowIter)) / 540ULL;
  colIter = static_cast<int>(threadId % 960ULL);
  threadId = (threadId - static_cast<unsigned long long>(colIter)) / 960ULL;
  rowBlockIter = static_cast<int>(threadId % 2ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowBlockIter)) / 2ULL;
  tmpVal = static_cast<int>(threadId);
  if ((tmpVal < 2) && (rowBlockIter < 2) && (colIter < 960) &&
      (rowIter < 540)) {
    unsigned int u1;
    tmpVal = tmpVal * 960 + 1;
    rowBlockIter = rowBlockIter * 540 + 1;
    u1 =
        frameLuma[((rowBlockIter + rowIter) + 1080 * ((tmpVal + colIter) - 1)) -
                  1] +
        1U;
    if (u1 > 65535U) {
      u1 = 65535U;
    }
    gpu_uint64_atomicAdd(
        &localHistogram
            [((static_cast<int>(u1) +
               ((static_cast<int>(
                     (static_cast<double>(rowBlockIter) - 1.0) / 540.0 + 1.0) -
                 1)
                << 12)) +
              ((static_cast<int>((static_cast<double>(tmpVal) - 1.0) / 960.0 +
                                 1.0) -
                1)
               << 13)) -
             1],
        1ULL);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long globalHistogram[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel17(
    unsigned long long globalHistogram[4096])
{
  unsigned long long threadId;
  int rowBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  rowBlockIter = static_cast<int>(threadId);
  if (rowBlockIter < 4096) {
    //  Local histograms are added to create the final global histogram
    globalHistogram[rowBlockIter] = 0ULL;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                unsigned long long localHistogram[16384]
//                unsigned long long globalHistogram[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel18(
    unsigned long long localHistogram[16384],
    unsigned long long globalHistogram[4096])
{
  unsigned long long threadId;
  int k;
  int rowBlockIter;
  int tmpVal;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId % 4096ULL);
  threadId = (threadId - static_cast<unsigned long long>(k)) / 4096ULL;
  rowBlockIter = static_cast<int>(threadId % 2ULL);
  threadId = (threadId - static_cast<unsigned long long>(rowBlockIter)) / 2ULL;
  tmpVal = static_cast<int>(threadId);
  if ((tmpVal < 2) && (rowBlockIter < 2) && (k < 4096)) {
    gpu_uint64_atomicAdd(
        &globalHistogram[k],
        localHistogram[(k + (rowBlockIter << 12)) + (tmpVal << 13)]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned long long globalHistogram[4096]
//                int *bin1Percent
//                int *bin99Percent
//                int *binVal
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void visionPipeline_kernel19(
    const unsigned long long globalHistogram[4096], int *bin1Percent,
    int *bin99Percent, int *binVal)
{
  unsigned long long threadId;
  int y;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  y = static_cast<int>(threadId);
  if (y < 2) {
    //  Dummy kernel invocation: This is a technique to keep the data on GPU
    //  while processing the loop with a single CUDA thread. This is a GPU Coder
    //  artifact.
    //  Dummy Kernel call
    while ((globalHistogram[*binVal] < 207360ULL) && (*binVal < 4096)) {
      (*binVal)++;
      *bin1Percent = *binVal;
    }
    *binVal = 4096;
    while (globalHistogram[*binVal - 1] >= 2052864ULL) {
      *bin99Percent = *binVal;
      (*binVal)--;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short inputFrame[2108160]
//                int *y
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel2(
    const unsigned short inputFrame[2108160], int *y)
{
  unsigned int blockStride;
  unsigned int m;
  unsigned int thBlkId;
  unsigned int threadId;
  unsigned int threadStride;
  int tmpRed0;
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<unsigned int>(mwGetThreadsPerBlock());
  if (static_cast<unsigned int>(mwGetBlockIndex()) == 19199U / blockStride) {
    m = 19199U % blockStride;
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 19198U) {
    tmpRed0 = inputFrame[(static_cast<int>(threadId) + 1) % 10 +
                         1098 * ((static_cast<int>(threadId) + 1) / 10)];
  }
  m = __ballot_sync(MAX_uint32_T, threadId <= 19198U);
  for (unsigned int idx{threadId + threadStride}; idx <= 19198U;
       idx += threadStride) {
    tmpRed0 += inputFrame[(static_cast<int>(idx) + 1) % 10 +
                          1098 * ((static_cast<int>(idx) + 1) / 10)];
  }
  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if ((threadId <= 19198U) && (thBlkId == 0U)) {
    atomicAdd(&y[0], tmpRed0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int *bin99Percent
//                const int *bin1Percent
//                double pixelGainLUT[4096]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel20(
    const int *bin99Percent, const int *bin1Percent, double pixelGainLUT[4096])
{
  unsigned long long threadId;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId);
  if (k < 4096) {
    int tmpVal;
    //  Gain lookup table for 12-bit intensity image
    tmpVal = k - *bin1Percent;
    if (tmpVal < 0) {
      tmpVal = 0;
    }
    pixelGainLUT[k] = static_cast<double>(tmpVal) * 4095.0 /
                      static_cast<double>(*bin99Percent - *bin1Percent);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double pixelGainLUT[4096]
//                const unsigned short stg3OutFrame[6220800]
//                unsigned short outFrame[6220800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel21(
    const double pixelGainLUT[4096], const unsigned short stg3OutFrame[6220800],
    unsigned short outFrame[6220800])
{
  unsigned long long threadId;
  int k;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId);
  if (k < 6220800) {
    double histCol;
    unsigned int u1;
    unsigned short u;
    //  Apply gain to all pixels of present frame
    //  Note: We can do an in-place operation instead of writing the output into
    //  new memory locations, but this destroys the original frame data, which
    //  could be later used to write to disk. This can be changed to in-place
    //  operation if needed.
    //  Note2: Conditional statements (if-else) create thread divergences. To
    //  avoid thread divergence, we transform the following if-else code
    //  patterns to addition statements Code:
    //    if (condition)
    //        outValue = statement_1;
    //    else
    //        outValue = statement_2;
    //    end
    //
    //  Optimization:
    //    outValue = (condition == true)*statement_1 + (condition ==
    //    false)*statement_2
    u = stg3OutFrame[k];
    u1 = (u == 0) + static_cast<unsigned int>((u > 0) * u);
    if (u1 > 65535U) {
      u1 = 65535U;
    }
    histCol = round(pixelGainLUT[static_cast<int>(u1) - 1]);
    if (histCol < 65536.0) {
      if (histCol >= 0.0) {
        u = static_cast<unsigned short>(histCol);
      } else {
        u = 0U;
      }
    } else if (histCol >= 65536.0) {
      u = MAX_uint16_T;
    } else {
      u = 0U;
    }
    u1 = static_cast<unsigned int>((u >= 4095) * 4095) +
         static_cast<unsigned int>((u < 4095) * u);
    if (u1 > 65535U) {
      u1 = 65535U;
    }
    outFrame[k] = static_cast<unsigned short>(u1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double meanBlackValue
//                const unsigned short inputFrame[2108160]
//                unsigned short frameLuma[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel3(
    const double meanBlackValue, const unsigned short inputFrame[2108160],
    unsigned short frameLuma[2073600])
{
  unsigned long long threadId;
  int k;
  int rowBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId % 1080ULL);
  rowBlockIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(k)) / 1080ULL);
  if ((rowBlockIter < 1920) && (k < 1080)) {
    int tmpVal;
    unsigned short u;
    tmpVal = static_cast<int>(
        round(static_cast<double>(inputFrame[(k + 1098 * rowBlockIter) + 18]) -
              meanBlackValue));
    if (tmpVal < 65536) {
      if (tmpVal >= 0) {
        u = static_cast<unsigned short>(tmpVal);
      } else {
        u = 0U;
      }
    } else {
      u = MAX_uint16_T;
    }
    frameLuma[k + 1080 * rowBlockIter] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short varargout_5[2073600]
//                const unsigned short varargout_4[2073600]
//                const unsigned short varargout_3[2073600]
//                const unsigned short varargout_2[2073600]
//                const unsigned short varargout_1[2073600]
//                unsigned short stg2OutFrame[6220800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel4(
    const unsigned short varargout_5[2073600],
    const unsigned short varargout_4[2073600],
    const unsigned short varargout_3[2073600],
    const unsigned short varargout_2[2073600],
    const unsigned short varargout_1[2073600],
    unsigned short stg2OutFrame[6220800])
{
  unsigned long long threadId;
  int b_rowBlockIter;
  int colIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  b_rowBlockIter = static_cast<int>(threadId % 540ULL);
  colIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(b_rowBlockIter)) / 540ULL);
  if ((colIter < 960) && (b_rowBlockIter < 540)) {
    int rowBlockIter;
    rowBlockIter = (colIter << 1) + 1;
    colIter = (b_rowBlockIter << 1) + 1;
    stg2OutFrame[(colIter + 1080 * (rowBlockIter - 1)) - 1] =
        varargout_1[(colIter + 1080 * (rowBlockIter - 1)) - 1];
    stg2OutFrame[(colIter + 1080 * (rowBlockIter - 1)) + 2073599] =
        varargout_2[(colIter + 1080 * (rowBlockIter - 1)) - 1];
    stg2OutFrame[(colIter + 1080 * (rowBlockIter - 1)) + 4147199] =
        varargout_3[(colIter + 1080 * (rowBlockIter - 1)) - 1];
    stg2OutFrame[(colIter + 1080 * rowBlockIter) - 1] =
        varargout_4[(colIter + 1080 * rowBlockIter) - 1];
    stg2OutFrame[(colIter + 1080 * rowBlockIter) + 2073599] =
        varargout_1[(colIter + 1080 * rowBlockIter) - 1];
    stg2OutFrame[(colIter + 1080 * rowBlockIter) + 4147199] =
        varargout_5[(colIter + 1080 * rowBlockIter) - 1];
    stg2OutFrame[colIter + 1080 * (rowBlockIter - 1)] =
        varargout_5[colIter + 1080 * (rowBlockIter - 1)];
    stg2OutFrame[(colIter + 1080 * (rowBlockIter - 1)) + 2073600] =
        varargout_1[colIter + 1080 * (rowBlockIter - 1)];
    stg2OutFrame[(colIter + 1080 * (rowBlockIter - 1)) + 4147200] =
        varargout_4[colIter + 1080 * (rowBlockIter - 1)];
    stg2OutFrame[colIter + 1080 * rowBlockIter] =
        varargout_3[colIter + 1080 * rowBlockIter];
    stg2OutFrame[(colIter + 1080 * rowBlockIter) + 2073600] =
        varargout_2[colIter + 1080 * rowBlockIter];
    stg2OutFrame[(colIter + 1080 * rowBlockIter) + 4147200] =
        varargout_1[colIter + 1080 * rowBlockIter];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short stg2OutFrame[6220800]
//                unsigned short frameLuma[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel5(
    const unsigned short stg2OutFrame[6220800],
    unsigned short frameLuma[2073600])
{
  unsigned long long threadId;
  int k;
  int rowBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId % 1080ULL);
  rowBlockIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(k)) / 1080ULL);
  if ((rowBlockIter < 1920) && (k < 1080)) {
    frameLuma[k + 1080 * rowBlockIter] = stg2OutFrame[k + 1080 * rowBlockIter];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short stg2OutFrame[6220800]
//                unsigned short frameLuma[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel6(
    const unsigned short stg2OutFrame[6220800],
    unsigned short frameLuma[2073600])
{
  unsigned long long threadId;
  int k;
  int rowBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId % 1080ULL);
  rowBlockIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(k)) / 1080ULL);
  if ((rowBlockIter < 1920) && (k < 1080)) {
    frameLuma[k + 1080 * rowBlockIter] =
        stg2OutFrame[(k + 1080 * rowBlockIter) + 2073600];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short stg2OutFrame[6220800]
//                unsigned short frameLuma[2073600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel7(
    const unsigned short stg2OutFrame[6220800],
    unsigned short frameLuma[2073600])
{
  unsigned long long threadId;
  int k;
  int rowBlockIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int>(threadId % 1080ULL);
  rowBlockIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(k)) / 1080ULL);
  if ((rowBlockIter < 1920) && (k < 1080)) {
    frameLuma[k + 1080 * rowBlockIter] =
        stg2OutFrame[(k + 1080 * rowBlockIter) + 4147200];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned short whitePatch[518400]
//                float gMat[129600]
//                float bMat[129600]
//                float rMat[129600]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void visionPipeline_kernel8(
    const unsigned short whitePatch[518400], float gMat[129600],
    float bMat[129600], float rMat[129600])
{
  unsigned long long threadId;
  int b_rowBlockIter;
  int colIter;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  b_rowBlockIter = static_cast<int>(threadId % 270ULL);
  colIter = static_cast<int>(
      (threadId - static_cast<unsigned long long>(b_rowBlockIter)) / 270ULL);
  if ((colIter < 480) && (b_rowBlockIter < 270)) {
    int k;
    int rowBlockIter;
    unsigned int u1;
    rowBlockIter = (colIter << 1) + 1;
    colIter = (b_rowBlockIter << 1) + 1;
    b_rowBlockIter =
        static_cast<int>(floor(static_cast<double>(colIter) / 2.0));
    k = static_cast<int>(floor(static_cast<double>(rowBlockIter) / 2.0));
    rMat[k + 270 * b_rowBlockIter] =
        whitePatch[(colIter + 540 * (rowBlockIter - 1)) - 1];
    bMat[k + 270 * b_rowBlockIter] = whitePatch[colIter + 540 * rowBlockIter];
    u1 = static_cast<unsigned int>(
             whitePatch[(colIter + 540 * rowBlockIter) - 1]) +
         whitePatch[colIter + 540 * (rowBlockIter - 1)];
    if (u1 > 65535U) {
      u1 = 65535U;
    }
    gMat[k + 270 * b_rowBlockIter] = static_cast<float>(u1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float rMat[129600]
//                float *b
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void visionPipeline_kernel9(
    const float rMat[129600], float *b)
{
  unsigned long long threadId;
  int indV;
  threadId =
      static_cast<unsigned long long>(mwGetGlobalThreadIndexInXDimension());
  indV = static_cast<int>(threadId);
  if (indV < 1) {
    *b = rMat[129599];
  }
}

//
// Arguments    : int val
//                unsigned int mask
//                unsigned int numActiveWarps
// Return Type  : int
//
static __device__ int workGroupReduction(int val, unsigned int mask,
                                         unsigned int numActiveWarps)
{
  __shared__ int shared[32];
  unsigned int lane;
  unsigned int threadId;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  threadId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  lane = threadId % warpSize;
  threadId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[threadId] = val;
  }
  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (threadId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }
  return val;
}

//
// Arguments    : float val
//                unsigned int mask
//                unsigned int numActiveWarps
// Return Type  : float
//
static __device__ float workGroupReduction(float val, unsigned int mask,
                                           unsigned int numActiveWarps)
{
  __shared__ float shared[32];
  unsigned int lane;
  unsigned int threadId;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  threadId = static_cast<unsigned int>(mwGetThreadIndexWithinBlock());
  lane = threadId % warpSize;
  threadId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[threadId] = val;
  }
  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (threadId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }
  return val;
}

//
// Arguments    : const unsigned short inputFrame[2108160]
//                const unsigned short whitePatch[518400]
//                const double *gainFactor
//                unsigned short outFrame[6220800]
// Return Type  : void
//
void visionPipeline(const unsigned short inputFrame[2108160],
                    const unsigned short whitePatch[518400], const double *,
                    unsigned short outFrame[6220800])
{
  double(*gpu_x)[2073600];
  double(*gpu_pixelGainLUT)[4096];
  unsigned long long(*gpu_localHistogram)[16384];
  unsigned long long(*gpu_globalHistogram)[4096];
  float(*gpu_bMat)[129600];
  float(*gpu_gMat)[129600];
  float(*gpu_rMat)[129600];
  float b;
  float c;
  float d;
  float *b_gpu_tmp;
  float *c_gpu_tmp;
  float *gpu_gainBChannel;
  float *gpu_gainGChannel;
  float *gpu_gainRChannel;
  float *gpu_tmp;
  int bin1Percent;
  int bin99Percent;
  int binVal;
  int *gpu_bin1Percent;
  int *gpu_bin99Percent;
  int *gpu_binVal;
  int *gpu_y;
  unsigned short(*gpu_stg2OutFrame)[6220800];
  unsigned short(*gpu_stg3OutFrame)[6220800];
  unsigned short(*gpu_frameLuma)[2073600];
  unsigned short(*gpu_varargout_1)[2073600];
  unsigned short(*gpu_varargout_2)[2073600];
  unsigned short(*gpu_varargout_3)[2073600];
  unsigned short(*gpu_varargout_4)[2073600];
  unsigned short(*gpu_varargout_5)[2073600];
  if (!isInitialized_gpuMEX) {
    visionPipeline_initialize();
  }
  mwCudaMalloc(&gpu_pixelGainLUT, 32768ULL);
  mwCudaMalloc(&gpu_binVal, 4ULL);
  mwCudaMalloc(&gpu_bin99Percent, 4ULL);
  mwCudaMalloc(&gpu_bin1Percent, 4ULL);
  mwCudaMalloc(&gpu_globalHistogram, 32768ULL);
  mwCudaMalloc(&gpu_localHistogram, 131072ULL);
  mwCudaMalloc(&gpu_x, 16588800ULL);
  mwCudaMalloc(&gpu_gainRChannel, 4ULL);
  mwCudaMalloc(&gpu_gainGChannel, 4ULL);
  mwCudaMalloc(&gpu_gainBChannel, 4ULL);
  mwCudaMalloc(&c_gpu_tmp, 4ULL);
  mwCudaMalloc(&b_gpu_tmp, 4ULL);
  mwCudaMalloc(&gpu_tmp, 4ULL);
  mwCudaMalloc(&gpu_rMat, 518400ULL);
  mwCudaMalloc(&gpu_bMat, 518400ULL);
  mwCudaMalloc(&gpu_gMat, 518400ULL);
  mwCudaMalloc(&gpu_stg3OutFrame, 12441600ULL);
  mwCudaMalloc(&gpu_stg2OutFrame, 12441600ULL);
  mwCudaMalloc(&gpu_varargout_5, 4147200ULL);
  mwCudaMalloc(&gpu_varargout_4, 4147200ULL);
  mwCudaMalloc(&gpu_varargout_3, 4147200ULL);
  mwCudaMalloc(&gpu_varargout_2, 4147200ULL);
  mwCudaMalloc(&gpu_varargout_1, 4147200ULL);
  mwCudaMalloc(&gpu_frameLuma, 4147200ULL);
  mwCudaMalloc(&gpu_y, 4ULL);
  //  Split Frame and Black Correction
  //  Constants:
  //  Rows for black correction cropping
  //  Rows for black level estimation
  //  Estimating the black value in both the frames
  visionPipeline_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(inputFrame,
                                                                  gpu_y);
  visionPipeline_kernel2<<<dim3(38U, 1U, 1U), dim3(512U, 1U, 1U)>>>(inputFrame,
                                                                    gpu_y);
  hipMemcpy(&binVal, gpu_y, 4ULL, hipMemcpyDeviceToHost);
  //  Subtract mean black and remove the crop the blackCorrection rows
  //  Debayer
  //  Function to call debayer algorithm on input image.
  //     %% Sizes and constants
  //  Debayer Function
  //  Convolve image with masks
  visionPipeline_kernel3<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      static_cast<double>(binVal) / 19200.0, inputFrame, *gpu_frameLuma);
  stencilKernel<<<8160U, 256U>>>(*gpu_frameLuma, 0U, *gpu_varargout_1,
                                 *gpu_varargout_2, *gpu_varargout_3,
                                 *gpu_varargout_4, *gpu_varargout_5);
  //  Plane Ordering
  //  Since stencil kernel performs
  //  BayerFormat = [(R-plane,G-plane,B-plane)->(Row coord, Col coord)]
  //  Following the RGGB bayer format
  visionPipeline_kernel4<<<dim3(1013U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_varargout_5, *gpu_varargout_4, *gpu_varargout_3, *gpu_varargout_2,
      *gpu_varargout_1, *gpu_stg2OutFrame);
  //  GRBG = [(4,1,5)->(1,1), (1,2,3)->(1,2); (3,2,1)->(2,1), (5,1,4)->(2,2)]
  //  outFrame = stg2OutFrame;
  //  Despeckle
  //  Despeckle Algorithm Caller
  //  Function handle to stencil kernel
  //  Despeckle Input Frame
  visionPipeline_kernel5<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_stg2OutFrame, *gpu_frameLuma);
  stencilKernel<<<8160U, 256U>>>(*gpu_frameLuma, 0U,
                                 *(unsigned short(*)[2073600]) &
                                     (*gpu_stg3OutFrame)[0]);
  visionPipeline_kernel6<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_stg2OutFrame, *gpu_frameLuma);
  stencilKernel<<<8160U, 256U>>>(*gpu_frameLuma, 0U,
                                 *(unsigned short(*)[2073600]) &
                                     (*gpu_stg3OutFrame)[2073600]);
  visionPipeline_kernel7<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_stg2OutFrame, *gpu_frameLuma);
  stencilKernel<<<8160U, 256U>>>(*gpu_frameLuma, 0U,
                                 *(unsigned short(*)[2073600]) &
                                     (*gpu_stg3OutFrame)[4147200]);
  //  White Balance
  //  Compute the mean of RGB channels in the bayer patch
  visionPipeline_kernel8<<<dim3(254U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      whitePatch, *gpu_gMat, *gpu_bMat, *gpu_rMat);
  visionPipeline_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_rMat,
                                                                  gpu_tmp);
  coder_reduce0<<<dim3(254U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_rMat, gpu_tmp);
  hipMemcpy(&b, gpu_tmp, 4ULL, hipMemcpyDeviceToHost);
  visionPipeline_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_gMat,
                                                                   b_gpu_tmp);
  coder_reduce1<<<dim3(254U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_gMat,
                                                            b_gpu_tmp);
  hipMemcpy(&c, b_gpu_tmp, 4ULL, hipMemcpyDeviceToHost);
  visionPipeline_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_bMat,
                                                                   c_gpu_tmp);
  coder_reduce2<<<dim3(254U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_bMat,
                                                            c_gpu_tmp);
  hipMemcpy(&d, c_gpu_tmp, 4ULL, hipMemcpyDeviceToHost);
  //  Create dummy kernel to keep the data on GPU
  visionPipeline_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      d / 518400.0F, 1.2F * c / 518400.0F, b / 518400.0F, gpu_gainBChannel,
      gpu_gainGChannel, gpu_gainRChannel);
  //  Luma Gain
  //  [stg5OutFrame, gainFactor] = stg_lumaGain(stg4OutFrame, gainFactor);
  //  GPU Pragmas
  //  Compute sizes and declaring constants
  //  Compute Luminance
  visionPipeline_kernel13<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      gpu_gainBChannel, gpu_gainGChannel, gpu_gainRChannel, *gpu_x,
      *gpu_stg3OutFrame);
  visionPipeline_kernel14<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_frameLuma, *gpu_x);
  //  Histogram Computation
  //  GPU Codegen: Block-wise histogram computation
  //  Histogram per block is stored in the local histogram matrix
  visionPipeline_kernel15<<<dim3(32U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_localHistogram);
  visionPipeline_kernel16<<<dim3(4050U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_frameLuma, *gpu_localHistogram);
  //  Local histograms are added to create the final global histogram
  visionPipeline_kernel17<<<dim3(8U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_globalHistogram);
  visionPipeline_kernel18<<<dim3(32U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_localHistogram, *gpu_globalHistogram);
  //  Histogram equalization
  //  Cumulative histogram values
  callThrustScan1D(&(*gpu_globalHistogram)[0], false, 4096);
  //  Identify 90-th percentile bin and computing the smoothing factor
  binVal = 0;
  bin1Percent = 0;
  bin99Percent = 0;
  //  Dummy kernel invocation: This is a technique to keep the data on GPU
  //  while processing the loop with a single CUDA thread. This is a GPU Coder
  //  artifact.
  hipMemcpy(gpu_bin1Percent, &bin1Percent, 4ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_bin99Percent, &bin99Percent, 4ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_binVal, &binVal, 4ULL, hipMemcpyHostToDevice);
  visionPipeline_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_globalHistogram, gpu_bin1Percent, gpu_bin99Percent, gpu_binVal);
  //  Gain lookup table for 12-bit intensity image
  visionPipeline_kernel20<<<dim3(8U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      gpu_bin99Percent, gpu_bin1Percent, *gpu_pixelGainLUT);
  //  Apply gain to all pixels of present frame
  //  Note: We can do an in-place operation instead of writing the output into
  //  new memory locations, but this destroys the original frame data, which
  //  could be later used to write to disk. This can be changed to in-place
  //  operation if needed.
  //  Note2: Conditional statements (if-else) create thread divergences. To
  //  avoid thread divergence, we transform the following if-else code patterns
  //  to addition statements
  //  Code:
  //    if (condition)
  //        outValue = statement_1;
  //    else
  //        outValue = statement_2;
  //    end
  //
  //  Optimization:
  //    outValue = (condition == true)*statement_1 + (condition ==
  //    false)*statement_2
  visionPipeline_kernel21<<<dim3(12150U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_pixelGainLUT, *gpu_stg3OutFrame, outFrame);
  //  RGB to YCbCr
  //  stg6OutFrame = stg_rgbToYCbCrConversion(stg5OutFrame);
  //  Image Sharpening
  //  outFrame = stg_sharpenImage(stg6OutFrame);
  mwCudaFree(gpu_y);
  mwCudaFree(&(*gpu_frameLuma)[0]);
  mwCudaFree(&(*gpu_varargout_1)[0]);
  mwCudaFree(&(*gpu_varargout_2)[0]);
  mwCudaFree(&(*gpu_varargout_3)[0]);
  mwCudaFree(&(*gpu_varargout_4)[0]);
  mwCudaFree(&(*gpu_varargout_5)[0]);
  mwCudaFree(&(*gpu_stg2OutFrame)[0]);
  mwCudaFree(&(*gpu_stg3OutFrame)[0]);
  mwCudaFree(&(*gpu_gMat)[0]);
  mwCudaFree(&(*gpu_bMat)[0]);
  mwCudaFree(&(*gpu_rMat)[0]);
  mwCudaFree(gpu_tmp);
  mwCudaFree(b_gpu_tmp);
  mwCudaFree(c_gpu_tmp);
  mwCudaFree(gpu_gainBChannel);
  mwCudaFree(gpu_gainGChannel);
  mwCudaFree(gpu_gainRChannel);
  mwCudaFree(&(*gpu_x)[0]);
  mwCudaFree(&(*gpu_localHistogram)[0]);
  mwCudaFree(&(*gpu_globalHistogram)[0]);
  mwCudaFree(gpu_bin1Percent);
  mwCudaFree(gpu_bin99Percent);
  mwCudaFree(gpu_binVal);
  mwCudaFree(&(*gpu_pixelGainLUT)[0]);
}

//
// File trailer for visionPipeline.cu
//
// [EOF]
//
