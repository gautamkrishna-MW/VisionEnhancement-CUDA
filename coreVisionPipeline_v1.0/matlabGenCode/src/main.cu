//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: main.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 13-Feb-2023 14:42:18
//

/*************************************************************************/
/* This automatically generated example CUDA main file shows how to call */
/* entry-point functions that MATLAB Coder generated. You must customize */
/* this file for your application. Do not modify this file directly.     */
/* Instead, make a copy of this file, modify it, and integrate it into   */
/* your development environment.                                         */
/*                                                                       */
/* This file initializes entry-point function arguments to a default     */
/* size and value before calling the entry-point functions. It does      */
/* not store or use any values returned from the entry-point functions.  */
/* If necessary, it does pre-allocate memory for returned values.        */
/* You can use this file as a starting point for a main function that    */
/* you can deploy in your application.                                   */
/*                                                                       */
/* After you copy the file, and before you deploy it, you must make the  */
/* following changes:                                                    */
/* * For variable-size function arguments, change the example sizes to   */
/* the sizes that your application requires.                             */
/* * Change the example values of function arguments to the values that  */
/* your application requires.                                            */
/* * If the entry-point functions return values, store these values or   */
/* otherwise use them as required by your application.                   */
/*                                                                       */
/*************************************************************************/

// Include Files
#include "main.h"
#include "visionPipeline.h"
#include "visionPipeline_terminate.h"

#include "time.h";
#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
using namespace cv;

#include <chrono>
#include <iostream>
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;


// Function Declarations
static void argInit_1098x1920_uint16_T(unsigned short result[2108160]);

static void argInit_540x960_uint16_T(unsigned short result[518400]);

static double argInit_real_T();

static unsigned short argInit_uint16_T();

// Function Definitions
//
// Arguments    : unsigned short result[2108160]
// Return Type  : void
//
static void argInit_1098x1920_uint16_T(unsigned short result[2108160])
{
  // Loop over the array to initialize each element.
  for (int i{0}; i < 2108160; i++) {
    // Set the value of the array element.
    // Change this value to the value that the application requires.
    result[i] = argInit_uint16_T();
  }
}

//
// Arguments    : unsigned short result[518400]
// Return Type  : void
//
static void argInit_540x960_uint16_T(unsigned short result[518400])
{
  // Loop over the array to initialize each element.
  for (int i{0}; i < 518400; i++) {
    // Set the value of the array element.
    // Change this value to the value that the application requires.
    result[i] = argInit_uint16_T();
  }
}

//
// Arguments    : void
// Return Type  : double
//
static double argInit_real_T()
{
  return 0.0;
}

//
// Arguments    : void
// Return Type  : unsigned short
//
static unsigned short argInit_uint16_T()
{
  return (unsigned short)rand() % 4096;
}

//
// Arguments    : int argc
//                char **argv
// Return Type  : int
//
int main(int, char **)
{
  // The initialize function is being called automatically from your entry-point
  // function. So, a call to initialize is not included here. Invoke the
  // entry-point functions.
  // You can call entry-point functions multiple times.
  main_visionPipeline();
  // Terminate the application.
  // You do not need to do this more than one time.
  visionPipeline_terminate();
  return 0;
}

//
// Arguments    : void
// Return Type  : void
//
void main_visionPipeline()
{
	/*static unsigned short outFrameLeft[6220800];
	static unsigned short outFrameRight[6220800];
	static unsigned short b[4216320];
	static unsigned short c[518400];*/

	unsigned short* outFrame;
	unsigned short* b;
	unsigned short* c;

	unsigned short* dev_outFrame;
	unsigned short* dev_b;
	unsigned short* dev_c;

	double gainFactor = 340.0;

	outFrame = (unsigned short*)malloc(6220800 * sizeof(unsigned short));
	
	hipMalloc((void**)&dev_b, 4216320 * sizeof(unsigned short));
	hipMalloc((void**)&dev_c, 518400 * sizeof(unsigned short));
	hipMalloc((void**)&dev_outFrame, 6220800 * sizeof(unsigned short));
	
	int tmpPtr = 15;
	int *dev_tmpPtr = nullptr;
	hipMalloc((void**)&dev_tmpPtr, 1 * sizeof(int));
	hipMemcpy(dev_tmpPtr, &tmpPtr, 1 * sizeof(int), hipMemcpyHostToDevice);

	char fileName[100] = { 0 };

	for (int i = 0; i < 45; i++)
	{
#if 1  
		sprintf(fileName, "/home/brain/gautamCodes/visionPipeline/datasetImages/flowerFrames/%0.4d.png", i + 1);
		char whitePatchFile[] = "/home/brain/gautamCodes/visionPipeline/datasetImages/flowerFrames/whitePatch.png";
#else
	
		sprintf(fileName, "/home/brain/gautamCodes/visionPipeline/datasetImages/lymphNode/%0.4d.png", i + 1);
		char whitePatchFile[] = "/home/brain/gautamCodes/visionPipeline/datasetImages/lymphNode/whitePatch.png";
#endif
		Mat inpImg = imread(fileName, IMREAD_ANYDEPTH);
		Mat inpPatch = imread(whitePatchFile, IMREAD_ANYDEPTH);
		
		imshow("OrgImg", inpImg.t() * 16);
		waitKey(10);

		b = (unsigned short*)inpImg.data;
		c = (unsigned short*)inpImg.data;
		
		/* Timings and Code Execution */
		auto tStart = high_resolution_clock::now();

		hipMemcpy(dev_b, b, 4216320 * sizeof(unsigned short), hipMemcpyHostToDevice);
		hipMemcpy(dev_c, c, 518400 * sizeof(unsigned short), hipMemcpyHostToDevice);
		
		auto tChkPt1 = high_resolution_clock::now();

		visionPipeline(dev_b, dev_c, &gainFactor, dev_outFrame);

		auto tChkPt2 = high_resolution_clock::now();

		hipMemcpy(outFrame, dev_outFrame, 6220800 * sizeof(unsigned short), hipMemcpyDeviceToHost);
		
		auto tStop = high_resolution_clock::now();

		auto memcpyH2D = duration_cast<milliseconds>(tChkPt1 - tStart);
		auto funExec = duration_cast<milliseconds>(tChkPt2 - tChkPt1);
		auto memcpyD2H = duration_cast<milliseconds>(tStop - tChkPt2);

		std::cout << "Memcpy H2D Time: " << memcpyH2D.count() << "ms\n";
		std::cout << "Func Exec Time: " << funExec.count() << "ms\n";
		std::cout << "Memcpy D2H Time: " << memcpyD2H.count() << "ms\n";
		std::cout << "Total Time: " << memcpyD2H.count() + funExec.count() + memcpyH2D.count() << "ms\n";
		std::cout << "\n\n" << std::endl;

		/* Display Frame */
		Mat outMat;
		Mat frameRGB[3];
		frameRGB[2] = Mat(1920, 1080, CV_16U, outFrame);
		frameRGB[1] = Mat(1920, 1080, CV_16U, outFrame + (1920 * 1080));
		frameRGB[0] = Mat(1920, 1080, CV_16U, outFrame + (1920 * 1080 * 2));
		merge(frameRGB, 3, outMat);
		imshow("OutMat", outMat.t() * 16);
		waitKey(10);
	}
}

//
// File trailer for main.cu
//
// [EOF]
//
