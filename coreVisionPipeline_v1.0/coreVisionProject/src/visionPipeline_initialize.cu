//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: visionPipeline_initialize.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 17-Feb-2023 18:05:57
//

// Include Files
#include "visionPipeline_initialize.h"
#include "visionPipeline_data.h"
#include "MWMemoryManager.hpp"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void visionPipeline_initialize()
{
  mwMemoryManagerInit(256U, 0U, 8U, 2048U);
  hipGetLastError();
  isInitialized_gpuMEX = true;
}

//
// File trailer for visionPipeline_initialize.cu
//
// [EOF]
//
