//
// Trial License - for use to evaluate programs for possible purchase as
// an end-user only.
// File: stg_chkImageSaturation_terminate.cu
//
// GPU Coder version                    : 2.4
// CUDA/C/C++ source code generated on  : 21-Feb-2023 16:48:30
//

// Include Files
#include "stg_chkImageSaturation_terminate.h"
#include "stg_chkImageSaturation_data.h"
#include "MWMemoryManager.hpp"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void stg_chkImageSaturation_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorString(errCode),
            hipGetErrorName(errCode));
    exit(errCode);
  }
  mwMemoryManagerTerminate();
  isInitialized_stg_chkImageSaturation = false;
}

//
// File trailer for stg_chkImageSaturation_terminate.cu
//
// [EOF]
//
